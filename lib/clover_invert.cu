#include "hip/hip_runtime.h"
#include <tune_quda.h>
#include <clover_field_order.h>
#include <complex_quda.h>

namespace quda {

  template <typename Clover>
  struct CloverInvertArg {
    const Clover clover;
    Clover inverse;
    double *trlogA;
//extra attributes for twisted mass clover
    bool twist;
    double mu2;
    CloverInvertArg(Clover &inverse, const Clover &clover) :
      inverse(inverse), clover(clover), twist(clover.Twisted()), mu2(clover.Mu2()){ }
  };

  /**
     Use a Cholesky decomposition to invert the clover matrix
     Here we use an inplace inversion which hopefully reduces register pressure
   */
  // FIXME - compute the trlog in this kernel
  template <typename Float, typename Clover>
  __device__ __host__ void cloverInvertCompute(CloverInvertArg<Clover> arg, int x, int parity) {

    Float A[72];
    double trlogA = 0.0; // fixme - write this out

    // load the clover term into memory
    arg.clover.load(A, x, parity);

    for (int ch=0; ch<2; ch++) {

      Float diag[6];
      Float tmp[6]; // temporary storage
      complex<Float> tri[15];

      // hack into the right order as MILC just to copy algorithm directly
      // FIXME use native ordering in the Cholseky 
      // factor of two is inherent to QUDA clover storage
      for (int i=0; i<6; i++) diag[i] = 2.0*A[ch*36+i];
      for (int i=0; i<2; i++) tri[i] = complex<Float>(2.0*A[ch*36+6+2*i], 2.0*A[ch*36+6+2*i+1]);
      tri[2] = complex<Float>(2.0*A[ch*36+6+2*5], 2.0*A[ch*36+6+2*5+1]);
      tri[3] = complex<Float>(2.0*A[ch*36+6+2*2], 2.0*A[ch*36+6+2*2+1]);
      tri[4] = complex<Float>(2.0*A[ch*36+6+2*6], 2.0*A[ch*36+6+2*6+1]);
      tri[5] = complex<Float>(2.0*A[ch*36+6+2*9], 2.0*A[ch*36+6+2*9+1]);
      tri[6] = complex<Float>(2.0*A[ch*36+6+2*3], 2.0*A[ch*36+6+2*3+1]);
      tri[7] = complex<Float>(2.0*A[ch*36+6+2*7], 2.0*A[ch*36+6+2*7+1]);
      tri[8] = complex<Float>(2.0*A[ch*36+6+2*10], 2.0*A[ch*36+6+2*10+1]);
      tri[9] = complex<Float>(2.0*A[ch*36+6+2*12], 2.0*A[ch*36+6+2*12+1]);
      tri[10] = complex<Float>(2.0*A[ch*36+6+2*4], 2.0*A[ch*36+6+2*4+1]);
      tri[11] = complex<Float>(2.0*A[ch*36+6+2*8], 2.0*A[ch*36+6+2*8+1]);
      tri[12] = complex<Float>(2.0*A[ch*36+6+2*11], 2.0*A[ch*36+6+2*11+1]);
      for (int i=13; i<15; i++) tri[i] = complex<Float>(2.0*A[ch*36+6+2*i], 2.0*A[ch*36+6+2*i+1]);
//Compute (T^2 + mu2) first, then invert (not optimized!):
      if(arg.twist)
      {
         //complex<Float> aux[15];//hmmm, better to reuse A-regs...
         //another solution just to define (but compiler may not be happy with this, swapping everything in
         //the global buffer):
         complex<Float>* aux = (complex<Float>*)A[ch*36];
         //compute off-diagonal terms:
//
         aux[ 0] = tri[0]*diag[0]+diag[1]*tri[0]+conj(tri[2])*tri[1]+conj(tri[4])*tri[3]+conj(tri[7])*tri[6]+conj(tri[11])*tri[10];
//
         aux[ 1] = tri[1]*diag[0]+diag[2]*tri[1]+tri[2]*tri[0]+conj(tri[5])*tri[3]+conj(tri[8])*tri[6]+conj(tri[12])*tri[10];

         aux[ 2] = tri[2]*diag[1]+diag[2]*tri[2]+tri[1]*conj(tri[0])+conj(tri[5])*tri[4]+conj(tri[8])*tri[7]+conj(tri[12])*tri[11];
//
         aux[ 3] = tri[3]*diag[0]+diag[3]*tri[3]+tri[4]*tri[0]+tri[5]*tri[1]+conj(tri[9])*tri[6]+conj(tri[13])*tri[10];

         aux[ 4] = tri[4]*diag[1]+diag[3]*tri[4]+tri[3]*conj(tri[0])+tri[5]*tri[2]+conj(tri[9])*tri[7]+conj(tri[13])*tri[11];

         aux[ 5] = tri[5]*diag[2]+diag[3]*tri[5]+tri[3]*conj(tri[1])+tri[4]*conj(tri[2])+conj(tri[9])*tri[8]+conj(tri[13])*tri[12];
//
         aux[ 6] = tri[6]*diag[0]+diag[4]*tri[6]+tri[7]*tri[0]+tri[8]*tri[1]+tri[9]*tri[3]+conj(tri[14])*tri[10];

         aux[ 7] = tri[7]*diag[1]+diag[4]*tri[7]+tri[6]*conj(tri[0])+tri[8]*tri[2]+tri[9]*tri[4]+conj(tri[14])*tri[11];

         aux[ 8] = tri[8]*diag[2]+diag[4]*tri[8]+tri[6]*conj(tri[1])+tri[7]*conj(tri[2])+tri[9]*tri[5]+conj(tri[14])*tri[12];

         aux[ 9] = tri[9]*diag[3]+diag[4]*tri[9]+tri[6]*conj(tri[3])+tri[7]*conj(tri[4])+tri[8]*conj(tri[5])+conj(tri[14])*tri[13];
//
         aux[10] = tri[10]*diag[0]+diag[5]*tri[10]+tri[11]*tri[0]+tri[12]*tri[1]+tri[13]*tri[3]+tri[14]*tri[6];

         aux[11] = tri[11]*diag[1]+diag[5]*tri[11]+tri[10]*conj(tri[0])+tri[12]*tri[2]+tri[13]*tri[4]+tri[14]*tri[7];

         aux[12] = tri[12]*diag[2]+diag[5]*tri[12]+tri[10]*conj(tri[1])+tri[11]*conj(tri[2])+tri[13]*tri[5]+tri[14]*tri[8];

         aux[13] = tri[13]*diag[3]+diag[5]*tri[13]+tri[10]*conj(tri[3])+tri[11]*conj(tri[4])+tri[12]*conj(tri[5])+tri[14]*tri[9];

         aux[14] = tri[14]*diag[4]+diag[5]*tri[14]+tri[10]*conj(tri[6])+tri[11]*conj(tri[7])+tri[12]*conj(tri[8])+tri[13]*conj(tri[9]);


         //update diagonal elements:
         diag[0] = (Float)arg.mu2+diag[0]*diag[0]+norm(tri[ 0])+norm(tri[ 1])+norm(tri[ 3])+norm(tri[ 6])+norm(tri[10]);
         diag[1] = (Float)arg.mu2+diag[1]*diag[1]+norm(tri[ 0])+norm(tri[ 2])+norm(tri[ 4])+norm(tri[ 7])+norm(tri[11]); 
         diag[2] = (Float)arg.mu2+diag[2]*diag[2]+norm(tri[ 1])+norm(tri[ 2])+norm(tri[ 5])+norm(tri[ 8])+norm(tri[12]); 
         diag[3] = (Float)arg.mu2+diag[3]*diag[3]+norm(tri[ 3])+norm(tri[ 4])+norm(tri[ 5])+norm(tri[ 9])+norm(tri[13]); 
         diag[4] = (Float)arg.mu2+diag[4]*diag[4]+norm(tri[ 6])+norm(tri[ 7])+norm(tri[ 8])+norm(tri[ 9])+norm(tri[14]);
         diag[5] = (Float)arg.mu2+diag[5]*diag[5]+norm(tri[10])+norm(tri[11])+norm(tri[12])+norm(tri[13])+norm(tri[14]);

        //update off-diagonal elements:
         for(int i = 0; i < 15; i++) tri[i] = aux[i];
      }
//
      for (int j=0; j<6; j++) {
	diag[j] = sqrt(diag[j]);
	tmp[j] = 1.0 / diag[j];

	for (int k=j+1; k<6; k++) {
	  int kj = k*(k-1)/2+j;
	  tri[kj] *= tmp[j];
	}

	for(int k=j+1;k<6;k++){
	  int kj=k*(k-1)/2+j;
	  diag[k] -= (tri[kj] * conj(tri[kj])).real();
	  for(int l=k+1;l<6;l++){
	    int lj=l*(l-1)/2+j;
	    int lk=l*(l-1)/2+k;
	    tri[lk] -= tri[lj] * conj(tri[kj]);
	  }
	}	
      }
      
      /* Accumulate trlogA */
      for (int j=0;j<6;j++) trlogA += (double)2.0*log((double)(diag[j]));

      /* Now use forward and backward substitution to construct inverse */
      complex<Float> v1[6];
      for (int k=0;k<6;k++) {
	for(int l=0;l<k;l++) v1[l] = complex<Float>(0.0, 0.0);

	/* Forward substitute */
	v1[k] = complex<Float>(tmp[k], 0.0);
	for(int l=k+1;l<6;l++){
	  complex<Float> sum = complex<Float>(0.0, 0.0);
	  for(int j=k;j<l;j++){
	    int lj=l*(l-1)/2+j;		    
	    sum -= tri[lj] * v1[j];
	  }
	  v1[l] = sum * tmp[l];
	}
	
	/* Backward substitute */
	v1[5] = v1[5] * tmp[5];
	for(int l=4;l>=k;l--){
	  complex<Float> sum = v1[l];
	  for(int j=l+1;j<6;j++){
	    int jl=j*(j-1)/2+l;
	    sum -= conj(tri[jl]) * v1[j];
	  }
	  v1[l] = sum * tmp[l];
	}
	
	/* Overwrite column k */
	diag[k] = v1[k].real();
	for(int l=k+1;l<6;l++){
	  int lk=l*(l-1)/2+k;
	  tri[lk] = v1[l];
	}
      }

      for (int i=0; i<6; i++) A[ch*36+i] = 0.5 * diag[i];
      for (int i=0; i<2; i++) {
	A[ch*36+6+2*i] = 0.5 * tri[i].real(); A[ch*36+6+2*i+1] = 0.5 * tri[i].imag();
      }
      A[ch*36+6+2*5] = 0.5 * tri[2].real(); A[ch*36+6+2*5+1] = 0.5 * tri[2].imag();
      A[ch*36+6+2*2] = 0.5 * tri[3].real(); A[ch*36+6+2*2+1] = 0.5 * tri[3].imag();
      A[ch*36+6+2*6] = 0.5 * tri[4].real(); A[ch*36+6+2*6+1] = 0.5 * tri[4].imag();
      A[ch*36+6+2*9] = 0.5 * tri[5].real(); A[ch*36+6+2*9+1] = 0.5 * tri[5].imag();
      A[ch*36+6+2*3] = 0.5 * tri[6].real(); A[ch*36+6+2*3+1] = 0.5 * tri[6].imag();
      A[ch*36+6+2*7] = 0.5 * tri[7].real(); A[ch*36+6+2*7+1] = 0.5 * tri[7].imag();
      A[ch*36+6+2*10] = 0.5 * tri[8].real(); A[ch*36+6+2*10+1] = 0.5 * tri[8].imag();
      A[ch*36+6+2*12] = 0.5 * tri[9].real(); A[ch*36+6+2*12+1] = 0.5 * tri[9].imag();
      A[ch*36+6+2*4] = 0.5 * tri[10].real(); A[ch*36+6+2*4+1] = 0.5 * tri[10].imag();
      A[ch*36+6+2*8] = 0.5 * tri[11].real(); A[ch*36+6+2*8+1] = 0.5 * tri[11].imag();
      A[ch*36+6+2*11] = 0.5 * tri[12].real(); A[ch*36+6+2*11+1] = 0.5 * tri[12].imag();

      for (int i=13; i<15; i++) {
	A[ch*36+6+2*i] = 0.5 * tri[i].real(); A[ch*36+6+2*i+1] = 0.5 * tri[i].imag();
      }
    }	     

    // save the inverted matrix
    arg.inverse.save(A, x, parity);
  }

  template <typename Float, typename Clover>
  void cloverInvert(CloverInvertArg<Clover> arg) {  
    for (int parity=0; parity<2; parity++) {
      for (int x=0; x<arg.clover.volumeCB; x++) {
	cloverInvertCompute<Float>(arg, x, parity);
      }
    }
  }

  template <typename Float, typename Clover>
  __global__ void cloverInvertKernel(CloverInvertArg<Clover> arg) {  
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx >= 2*arg.clover.volumeCB) return;
    int parity = (idx >= arg.clover.volumeCB) ? 1 : 0;
    idx -= parity*arg.clover.volumeCB;
    
    cloverInvertCompute<Float>(arg, idx, parity);
  }

  template <typename Float, typename Clover>
  class CloverInvert : Tunable {
    CloverInvertArg<Clover> arg;
    const QudaFieldLocation location;

  private:
    unsigned int sharedBytesPerThread() const { return 0; }
    unsigned int sharedBytesPerBlock(const TuneParam &param) const { return 0 ;}

    bool tuneGridDim() const { return false; } // Don't tune the grid dimensions.
    unsigned int minThreads() const { return 2*arg.clover.volumeCB; }

  public:
    CloverInvert(CloverInvertArg<Clover> &arg, QudaFieldLocation location) 
      : arg(arg), location(location) { ; }
    virtual ~CloverInvert() { ; }
  
    void apply(const hipStream_t &stream) {
      TuneParam tp = tuneLaunch(*this, getTuning(), getVerbosity());
      if (location == QUDA_CUDA_FIELD_LOCATION) {
	cloverInvertKernel<Float, Clover> <<<tp.grid, tp.block, tp.shared_bytes, stream>>>(arg);
      } else {
	cloverInvert<Float, Clover>(arg);
      }
    }

    TuneKey tuneKey() const {
      std::stringstream vol, aux;
      vol << arg.clover.volumeCB; 
      aux << "stride=" << arg.clover.stride;
      return TuneKey(vol.str(), typeid(*this).name(), aux.str());
    }

    std::string paramString(const TuneParam &param) const { // Don't bother printing the grid dim.
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 0; } 
    long long bytes() const { return 2*arg.clover.volumeCB*(arg.inverse.Bytes() + arg.clover.Bytes()); } 
  };

  template <typename Float, typename Clover>
  void cloverInvert(Clover inverse, const Clover clover, QudaFieldLocation location) {
    CloverInvertArg<Clover> arg(inverse, clover);
    CloverInvert<Float,Clover> invert(arg, location);
    invert.apply(0);
  }

  template <typename Float>
 void cloverInvert(const CloverField &clover, QudaFieldLocation location) {
    if (clover.Order() == QUDA_FLOAT2_CLOVER_ORDER) {
      cloverInvert<Float>(FloatNOrder<Float,72,2>(clover, 1), 
			  FloatNOrder<Float,72,2>(clover, 0), location);
    } else if (clover.Order() == QUDA_FLOAT4_CLOVER_ORDER) {
      cloverInvert<Float>(FloatNOrder<Float,72,4>(clover, 1), 
			  FloatNOrder<Float,72,4>(clover, 0), location);
    } else if (clover.Order() == QUDA_PACKED_CLOVER_ORDER) {
      cloverInvert<Float>(QDPOrder<Float,72>(clover, 1), 
			  QDPOrder<Float,72>(clover, 0), location);
    } else if (clover.Order() == QUDA_QDPJIT_CLOVER_ORDER) {

#ifdef BUILD_QDPJIT_INTERFACE
      cloverInvert<Float>(QDPJITOrder<Float,72>(clover, 1), 
			  QDPJITOrder<Float,72>(clover, 0), location);
#else
      errorQuda("QDPJIT interface has not been built\n");
#endif

    } else if (clover.Order() == QUDA_BQCD_CLOVER_ORDER) {
      errorQuda("BQCD output not supported");
    } else {
      errorQuda("Clover field %d order not supported", clover.Order());
    }

  }

  // this is the function that is actually called, from here on down we instantiate all required templates
  void cloverInvert(CloverField &clover, QudaFieldLocation location) {
    if (clover.Precision() == QUDA_HALF_PRECISION && clover.Order() > 4) 
      errorQuda("Half precision not supported for order %d", clover.Order());
//ok, currently we clover is overwritten... so actually must be cloverInv...
    if (clover.Precision() == QUDA_DOUBLE_PRECISION) {
      cloverInvert<double>(clover, location);
    } else if (clover.Precision() == QUDA_SINGLE_PRECISION) {
      cloverInvert<float>(clover, location);
    } else {
      errorQuda("Precision %d not supported", clover.Precision());
    }
  }

} // namespace quda
