#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <string>
#include <iostream>

#include <color_spinor_field.h>
#include <clover_field.h>

// these control the Wilson-type actions
#ifdef GPU_WILSON_DIRAC
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR
//#define DIRECT_ACCESS_CLOVER
#endif // GPU_WILSON_DIRAC

//these are access control for staggered action
#ifdef GPU_STAGGERED_DIRAC
#if (__COMPUTE_CAPABILITY__ >= 300) // Kepler works best with texture loads only
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#elif (__COMPUTE_CAPABILITY__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
//#define DIRECT_ACCESS_ACCUM
//#define DIRECT_ACCESS_INTER
//#define DIRECT_ACCESS_PACK
#endif
#endif // GPU_STAGGERED_DIRAC

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>
#include <blas_quda.h>
#include <face_quda.h>

#include <inline_ptx.h>

enum KernelType {
  INTERIOR_KERNEL = 5,
  EXTERIOR_KERNEL_X = 0,
  EXTERIOR_KERNEL_Y = 1,
  EXTERIOR_KERNEL_Z = 2,
  EXTERIOR_KERNEL_T = 3
};

namespace quda {

  struct DslashParam {
    int threads; // the desired number of active threads
    int parity;  // Even-Odd or Odd-Even
    int commDim[QUDA_MAX_DIM]; // Whether to do comms or not
    int ghostDim[QUDA_MAX_DIM]; // Whether a ghost zone has been allocated for a given dimension
    int ghostOffset[QUDA_MAX_DIM+1];
    int ghostNormOffset[QUDA_MAX_DIM+1];
    KernelType kernel_type; //is it INTERIOR_KERNEL, EXTERIOR_KERNEL_X/Y/Z/T

#ifdef USE_TEXTURE_OBJECTS
    hipTextureObject_t inTex;
    hipTextureObject_t inTexNorm;
    hipTextureObject_t xTex;
    hipTextureObject_t xTexNorm;
    hipTextureObject_t outTex;
    hipTextureObject_t outTexNorm;
    hipTextureObject_t gauge0Tex; // also applies to fat gauge
    hipTextureObject_t gauge1Tex; // also applies to fat gauge
    hipTextureObject_t longGauge0Tex;
    hipTextureObject_t longGauge1Tex;
    hipTextureObject_t cloverTex;
    hipTextureObject_t cloverNormTex;
#endif
  };

  DslashParam dslashParam;

  // these are set in initDslashConst
  int Vspatial;

  static hipEvent_t packEnd[Nstream];
  static hipEvent_t gatherStart[Nstream];
  static hipEvent_t gatherEnd[Nstream];
  static hipEvent_t scatterStart[Nstream];
  static hipEvent_t scatterEnd[Nstream];
  static hipEvent_t dslashStart;
  static hipEvent_t dslashEnd;

  static struct timeval dslashStart_h;
#ifdef MULTI_GPU
  static struct timeval commsStart[Nstream];
  static struct timeval commsEnd[Nstream];
#endif

  // these events are only used for profiling
#ifdef DSLASH_PROFILING
#define DSLASH_TIME_PROFILE() dslashTimeProfile()

  static hipEvent_t packStart[Nstream];
  static hipEvent_t kernelStart[Nstream];
  static hipEvent_t kernelEnd[Nstream];

  // dimension 2 because we want absolute and relative
  float packTime[Nstream][2];
  float gatherTime[Nstream][2];
  float commsTime[Nstream][2];
  float scatterTime[Nstream][2];
  float kernelTime[Nstream][2];
  float dslashTime;
#define CUDA_EVENT_RECORD(a,b) hipEventRecord(a,b)
#else
#define CUDA_EVENT_RECORD(a,b)
#define DSLASH_TIME_PROFILE()
#endif

  static FaceBuffer *face;
  static cudaColorSpinorField *inSpinor;

  // For tuneLaunch() to uniquely identify a suitable set of launch parameters, we need copies of a few of
  // the constants set by initDslashConstants().
  static struct {
    int x[4];
    int Ls;
    unsigned long long VolumeCB() { return x[0]*x[1]*x[2]*x[3]/2; }
    // In the future, we may also want to add gauge_fixed, sp_stride, ga_stride, cl_stride, etc.
  } dslashConstants;

  // dslashTuning = QUDA_TUNE_YES enables autotuning when the dslash is
  // first launched
  static QudaTune dslashTuning = QUDA_TUNE_NO;
  static QudaVerbosity verbosity = QUDA_SILENT;

  void setDslashTuning(QudaTune tune, QudaVerbosity verbose)
  {
    dslashTuning = tune;
    verbosity = verbose;
  }

  // determines whether the temporal ghost zones are packed with a gather kernel,
  // as opposed to multiple calls to hipMemcpy()
  static bool kernelPackT = false;

  void setKernelPackT(bool packT) { kernelPackT = packT; }

  bool getKernelPackT() { return kernelPackT; }


#include <dslash_textures.h>
#include <dslash_constants.h>

#if defined(DIRECT_ACCESS_LINK) || defined(DIRECT_ACCESS_WILSON_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_ACCUM) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_INTER) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR) || \
  defined(DIRECT_ACCESS_CLOVER)

  static inline __device__ float short2float(short a) {
    return (float)a/MAX_SHORT;
  }

  static inline __device__ short float2short(float c, float a) {
    return (short)(a*c*MAX_SHORT);
  }

  static inline __device__ short4 float42short4(float c, float4 a) {
    return make_short4(float2short(c, a.x), float2short(c, a.y), float2short(c, a.z), float2short(c, a.w));
  }

  static inline __device__ float4 short42float4(short4 a) {
    return make_float4(short2float(a.x), short2float(a.y), short2float(a.z), short2float(a.w));
  }

  static inline __device__ float2 short22float2(short2 a) {
    return make_float2(short2float(a.x), short2float(a.y));
  }
#endif // DIRECT_ACCESS inclusions

  // Enable shared memory dslash for Fermi architecture
  //#define SHARED_WILSON_DSLASH
  //#define SHARED_8_BYTE_WORD_SIZE // 8-byte shared memory access

#include <pack_face_def.h>        // kernels for packing the ghost zones and general indexing
#include <staggered_dslash_def.h> // staggered Dslash kernels
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <dw_dslash_def.h>        // Domain Wall kernels
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <tm_core.h>              // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone
#include <tm_ndeg_dslash_def.h>   // Non-degenerate twisted Mass

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef CLOVER_SHARED_FLOATS_PER_THREAD
#define CLOVER_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef NDEGTM_SHARED_FLOATS_PER_THREAD
#define NDEGTM_SHARED_FLOATS_PER_THREAD 0
#endif


  void setFace(const FaceBuffer &Face) {
    face = (FaceBuffer*)&Face; // nasty
  }


  void createDslashEvents()
  {
#ifndef DSLASH_PROFILING
    // add hipEventDisableTiming for lower sync overhead
    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packEnd[i], hipEventDisableTiming);
      hipEventCreate(&gatherStart[i], hipEventDisableTiming);
      hipEventCreate(&gatherEnd[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterStart[i], hipEventDisableTiming);
      hipEventCreateWithFlags(&scatterEnd[i], hipEventDisableTiming);
    }
    hipEventCreateWithFlags(&dslashStart, hipEventDisableTiming);
    hipEventCreateWithFlags(&dslashEnd, hipEventDisableTiming);
#else
    hipEventCreate(&dslashStart);
    hipEventCreate(&dslashEnd);

    for (int i=0; i<Nstream; i++) {
      hipEventCreate(&packStart[i]);
      hipEventCreate(&packEnd[i]);

      hipEventCreate(&gatherStart[i]);
      hipEventCreate(&gatherEnd[i]);

      hipEventCreate(&scatterStart[i]);
      hipEventCreate(&scatterEnd[i]);

      hipEventCreate(&kernelStart[i]);
      hipEventCreate(&kernelEnd[i]);

      kernelTime[i][0] = 0.0;
      kernelTime[i][1] = 0.0;

      gatherTime[i][0] = 0.0;
      gatherTime[i][1] = 0.0;

      commsTime[i][0] = 0.0;
      commsTime[i][1] = 0.0;

      scatterTime[i][0] = 0.0;
      scatterTime[i][1] = 0.0;
    }
#endif

    checkCudaError();
  }


  void destroyDslashEvents()
  {
    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packEnd[i]);
      hipEventDestroy(gatherStart[i]);
      hipEventDestroy(gatherEnd[i]);
      hipEventDestroy(scatterStart[i]);
      hipEventDestroy(scatterEnd[i]);
    }

    hipEventDestroy(dslashStart);
    hipEventDestroy(dslashEnd);

#ifdef DSLASH_PROFILING
    for (int i=0; i<Nstream; i++) {
      hipEventDestroy(packStart[i]);
      hipEventDestroy(kernelStart[i]);
      hipEventDestroy(kernelEnd[i]);
    }
#endif

    checkCudaError();
  }


#define MORE_GENERIC_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...) \
  if (x==0) {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else {								\
      FUNC ## 8 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  } else {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
      FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  }

#ifndef MULTI_GPU

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  default:								\
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
  }

#else

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_X:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Y:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Z:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_T:						\
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  }

#endif

  // macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...)	\
  if (!dagger) {							\
    GENERIC_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      } else {								\
    GENERIC_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      }

  // macro used for staggered dslash
#define STAGGERED_DSLASH(gridDim, blockDim, shared, stream, param, ...)	\
  GENERIC_DSLASH(staggeredDslash, , Axpy, gridDim, blockDim, shared, stream, param, __VA_ARGS__)


#define MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...) \
  if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
    FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
    FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
    FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
  }									

#ifndef MULTI_GPU

#define GENERIC_ASYM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  default:								\
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
  }

#else

#define GENERIC_ASYM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_X:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Y:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Z:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_T:						\
    MORE_GENERIC_ASYM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  }

#endif

  // macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define ASYM_DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...) \
  if (!dagger) {							\
    GENERIC_ASYM_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      } else {								\
    GENERIC_ASYM_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      }



//macro used for twisted mass dslash:

#define MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...) \
  if (x == 0 && d == 0) {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## Twist ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## Twist ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else {								\
      FUNC ## 8 ## DAG ## Twist ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  } else if (x != 0 && d == 0) {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## Twist ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## Twist ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
      FUNC ## 8 ## DAG ## Twist ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  } else if (x == 0 && d != 0) {								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param); \
    } else {								\
      FUNC ## 8 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  } else{								\
    if (reconstruct == QUDA_RECONSTRUCT_NO) {				\
      FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {			\
      FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {			\
      FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }									\
  }

#ifndef MULTI_GPU

#define GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  default:								\
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type); \
  }

#else

#define GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...) \
  switch(param.kernel_type) {						\
  case INTERIOR_KERNEL:							\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_X:						\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Y:						\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_Z:						\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  case EXTERIOR_KERNEL_T:						\
    MORE_GENERIC_NDEG_TM_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      break;								\
  }

#endif

#define NDEG_TM_DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...)	\
  if (!dagger) {							\
    GENERIC_NDEG_TM_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      } else {								\
    GENERIC_NDEG_TM_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
      }
//end of tm dslash macro


  // Use an abstract class interface to drive the different CUDA dslash
  // kernels. All parameters are curried into the derived classes to
  // allow a simple interface.
  class DslashCuda : public Tunable {
  protected:
    cudaColorSpinorField *out;
    const cudaColorSpinorField *in;
    const cudaColorSpinorField *x;
    char *saveOut, *saveOutNorm;

    int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool advanceGridDim(TuneParam &param) const { return false; } // Don't tune the grid dimensions.
    bool advanceBlockDim(TuneParam &param) const {
      bool advance = Tunable::advanceBlockDim(param);
      if (advance) {
	param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
	if (param.grid.x > deviceProp.maxGridSize[0]) {
	  warningQuda("Autotuner is skipping blockDim=%u (gridDim=%u) because lattice volume is too large",
		      param.block.x, param.grid.x);
	  advance = advanceBlockDim(param);
	}
      }
      return advance;
    }

  public:
    DslashCuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
	       const cudaColorSpinorField *x) 
      : out(out), in(in), x(x), saveOut(0), saveOutNorm(0) { }
    virtual ~DslashCuda() { }
    virtual TuneKey tuneKey() const;
    std::string paramString(const TuneParam &param) const // Don't bother printing the grid dim.
    {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }
    virtual int Nface() { return 2; }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
      if (param.grid.x > deviceProp.maxGridSize[0]) {
	warningQuda("Autotuner is skipping blockDim=%u (gridDim=%u) because lattice volume is too large",
		    param.block.x, param.grid.x);
	bool ok = advanceBlockDim(param);
	if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
      }
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 1, 1);
      if (param.grid.x > deviceProp.maxGridSize[0]) {
	errorQuda("Lattice volume is too large for default blockDim");
      }
    }

    virtual void preTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	saveOut = new char[in->Bytes()];
	hipMemcpy(saveOut, out->V(), in->Bytes(), hipMemcpyDeviceToHost);
	if (out->Precision() == QUDA_HALF_PRECISION) {
	  saveOutNorm = new char[in->NormBytes()];
	  hipMemcpy(saveOutNorm, out->Norm(), in->NormBytes(), hipMemcpyDeviceToHost);
	}
      }
    }

    virtual void postTune()
    {
      if (dslashParam.kernel_type < 5) { // exterior kernel
	hipMemcpy(out->V(), saveOut, in->Bytes(), hipMemcpyHostToDevice);
	delete[] saveOut;
	if (out->Precision() == QUDA_HALF_PRECISION) {
	  hipMemcpy(out->Norm(), saveOutNorm, in->NormBytes(), hipMemcpyHostToDevice);
	  delete[] saveOutNorm;
	}
      }
    }

  };

  TuneKey DslashCuda::tuneKey() const
  {
    std::stringstream vol, aux;
  
    vol << dslashConstants.x[0] << "x";
    vol << dslashConstants.x[1] << "x";
    vol << dslashConstants.x[2] << "x";
    vol << dslashConstants.x[3];

    aux << "type=";
#ifdef MULTI_GPU
    char comm[5], ghost[5];
    switch (dslashParam.kernel_type) {
    case INTERIOR_KERNEL: aux << "interior"; break;
    case EXTERIOR_KERNEL_X: aux << "exterior_x"; break;
    case EXTERIOR_KERNEL_Y: aux << "exterior_y"; break;
    case EXTERIOR_KERNEL_Z: aux << "exterior_z"; break;
    case EXTERIOR_KERNEL_T: aux << "exterior_t"; break;
    }
    for (int i=0; i<4; i++) {
      comm[i] = (dslashParam.commDim[i] ? '1' : '0');
      ghost[i] = (dslashParam.ghostDim[i] ? '1' : '0');
    }
    comm[4] = '\0'; ghost[4] = '\0';
    aux << ",comm=" << comm;
    if (dslashParam.kernel_type == INTERIOR_KERNEL) {
      aux << ",ghost=" << ghost;
    }
#else
    aux << "single-GPU";
#endif // MULTI_GPU
    return TuneKey(vol.str(), typeid(*this).name(), aux.str());
  }

  /** This derived class is specifically for driving the Dslash kernels
      that use shared memory blocking.  This only applies on Fermi and
      upwards, and only for the interior kernels. */
#if (__COMPUTE_CAPABILITY__ >= 200 && defined(SHARED_WILSON_DSLASH)) 
  class SharedDslashCuda : public DslashCuda {
  protected:
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; } // FIXME: this isn't quite true, but works
    bool advanceSharedBytes(TuneParam &param) const { 
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::advanceSharedBytes(param);
      else return false;
    } // FIXME - shared memory tuning only supported on exterior kernels

    /** Helper function to set the shared memory size from the 3-d block size */
    int sharedBytes(const dim3 &block) const { 
      int warpSize = 32; // FIXME - query from device properties
      int block_xy = block.x*block.y;
      if (block_xy % warpSize != 0) block_xy = ((block_xy / warpSize) + 1)*warpSize;
      return block_xy*block.z*sharedBytesPerThread();
    }

    /** Helper function to set the 3-d grid size from the 3-d block size */
    dim3 createGrid(const dim3 &block) const {
      unsigned int gx = ((dslashConstants.x[0]/2)*dslashConstants.x[3] + block.x - 1) / block.x;
      unsigned int gy = (dslashConstants.x[1] + block.y - 1 ) / block.y;	
      unsigned int gz = (dslashConstants.x[2] + block.z - 1) / block.z;
      return dim3(gx, gy, gz);
    }

    /** Advance the 3-d block size. */
    bool advanceBlockDim(TuneParam &param) const {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::advanceBlockDim(param);
      const unsigned int min_threads = 2;
      const unsigned int max_threads = 512; // FIXME: use deviceProp.maxThreadsDim[0];
      const unsigned int max_shared = 16384*3; // FIXME: use deviceProp.sharedMemPerBlock;
    
      // set the x-block dimension equal to the entire x dimension
      bool set = false;
      dim3 blockInit = param.block;
      blockInit.z++;
      for (unsigned bx=blockInit.x; bx<=dslashConstants.x[0]/2; bx++) {
	//unsigned int gx = (dslashConstants.x[0]*dslashConstants.x[3] + bx - 1) / bx;
	for (unsigned by=blockInit.y; by<=dslashConstants.x[1]; by++) {
	  unsigned int gy = (dslashConstants.x[1] + by - 1 ) / by;	
	
	  if (by > 1 && (by%2) != 0) continue; // can't handle odd blocks yet except by=1
	
	  for (unsigned bz=blockInit.z; bz<=dslashConstants.x[2]; bz++) {
	    unsigned int gz = (dslashConstants.x[2] + bz - 1) / bz;
	  
	    if (bz > 1 && (bz%2) != 0) continue; // can't handle odd blocks yet except bz=1
	    if (bx*by*bz > max_threads) continue;
	    if (bx*by*bz < min_threads) continue;
	    // can't yet handle the last block properly in shared memory addressing
	    if (by*gy != dslashConstants.x[1]) continue;
	    if (bz*gz != dslashConstants.x[2]) continue;
	    if (sharedBytes(dim3(bx, by, bz)) > max_shared) continue;

	    param.block = dim3(bx, by, bz);	  
	    set = true; break;
	  }
	  if (set) break;
	  blockInit.z = 1;
	}
	if (set) break;
	blockInit.y = 1;
      }

      if (param.block.x > dslashConstants.x[0]/2 && param.block.y > dslashConstants.x[1] &&
	  param.block.z > dslashConstants.x[2] || !set) {
	//||sharedBytesPerThread()*param.block.x > max_shared) {
	param.block = dim3(dslashConstants.x[0]/2, 1, 1);
	return false;
      } else { 
	param.grid = createGrid(param.block);
	param.shared_bytes = sharedBytes(param.block);
	return true; 
      }
    
    }

  public:
    SharedDslashCuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x) : DslashCuda(out, in, x) { ; }
    virtual ~SharedDslashCuda() { ; }
    std::string paramString(const TuneParam &param) const // override and print out grid as well
    {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "grid=(" << param.grid.x << "," << param.grid.y << "," << param.grid.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    virtual void initTuneParam(TuneParam &param) const
    {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) return DslashCuda::initTuneParam(param);

      param.block = dim3(dslashConstants.x[0]/2, 1, 1);
      param.grid = createGrid(param.block);
      param.shared_bytes = sharedBytes(param.block);
    }

    /** Sets default values for when tuning is disabled - this is guaranteed to work, but will be slow */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      if (dslashParam.kernel_type != INTERIOR_KERNEL) DslashCuda::defaultTuneParam(param);
      else initTuneParam(param);
    }
  };
#else /** For pre-Fermi architectures */
  class SharedDslashCuda : public DslashCuda {
  public:
    SharedDslashCuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x) : DslashCuda(out, in, x) { }
    virtual ~SharedDslashCuda() { }
  };
#endif


  template <typename sFloat, typename gFloat>
  class WilsonDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const int dagger;
    const double a;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200) // Fermi uses shared memory for common input
      if (dslashParam.kernel_type == INTERIOR_KERNEL) { // Interior kernels use shared memory for common iunput
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else { // Exterior kernels use no shared memory
	return 0;
      }
#else // Pre-Fermi uses shared memory only for pseudo-registers
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }

  public:
    WilsonDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
		     const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
		     const cudaColorSpinorField *x, const double a, const int dagger)
      : SharedDslashCuda(out, in, x), gauge0(gauge0), gauge1(gauge1), 
	reconstruct(reconstruct), dagger(dagger), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x); 
    }

    virtual ~WilsonDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      DSLASH(dslash, tp.grid, tp.block, tp.shared_bytes, stream, 
	     dslashParam, (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	     (sFloat*)in->V(), (float*)in->Norm(), (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
    }

    long long flops() const { return (x ? 1368ll : 1320ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };

  template <typename sFloat, typename gFloat, typename cFloat>
  class CloverDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const cFloat *clover;
    const float *cloverNorm;
    const int dagger;
    const double a;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200)
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
#else
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }
  public:
    CloverDslashCuda(cudaColorSpinorField *out,  const gFloat *gauge0, const gFloat *gauge1, 
		     const QudaReconstructType reconstruct, const cFloat *clover, 
		     const float *cloverNorm, const cudaColorSpinorField *in, 
		     const cudaColorSpinorField *x, const double a, const int dagger)
      : SharedDslashCuda(out, in, x), gauge0(gauge0), gauge1(gauge1), clover(clover),
	cloverNorm(cloverNorm), reconstruct(reconstruct), dagger(dagger), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x); 
    }
    virtual ~CloverDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      DSLASH(cloverDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, clover, cloverNorm, 
	     (sFloat*)in->V(), (float*)in->Norm(), (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
    }

    long long flops() const { return (x ? 1872ll : 1824ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };

  template <typename sFloat, typename gFloat, typename cFloat>
  class AsymCloverDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const cFloat *clover;
    const float *cloverNorm;
    const int dagger;
    const double a;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200)
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
	int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
	return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
      } else {
	return 0;
      }
#else
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return DSLASH_SHARED_FLOATS_PER_THREAD * reg_size;
#endif
    }

  public:
    AsymCloverDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
			 const QudaReconstructType reconstruct, const cFloat *clover, 
			 const float *cloverNorm, const cudaColorSpinorField *in,
			 const cudaColorSpinorField *x, const double a, const int dagger)
      : SharedDslashCuda(out, in, x), gauge0(gauge0), gauge1(gauge1), clover(clover),
	cloverNorm(cloverNorm), reconstruct(reconstruct), dagger(dagger), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
      if (!x) errorQuda("Asymmetric clover dslash only defined for Xpay");
    }
    virtual ~AsymCloverDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str() + ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
	errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      ASYM_DSLASH(asymCloverDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
		  (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, clover, cloverNorm, 
		  (sFloat*)in->V(), (float*)in->Norm(), (sFloat*)x, (float*)x->Norm(), a);
    }

    long long flops() const { return 1872ll * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };

  void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		     const int dagger, const QudaTwistGamma5Type twist) {
    if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
      a = 2.0 * kappa * mu;
      b = 1.0;
    } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
      a = -2.0 * kappa * mu;
      b = 1.0 / (1.0 + a*a);
    } else {
      errorQuda("Twist type %d not defined\n", twist);
    }
    if (dagger) a *= -1.0;

  }

  template <typename sFloat, typename gFloat>
  class TwistedDslashCuda : public SharedDslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const int dagger;
    double a, b, c, d;

  protected:
    int sharedBytesPerThread() const
    {
#if (__COMPUTE_CAPABILITY__ >= 200)
      if (dslashParam.kernel_type == INTERIOR_KERNEL) {
        int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
        return ((in->TwistFlavor() == QUDA_TWIST_PLUS || in->TwistFlavor() == QUDA_TWIST_MINUS) ? DSLASH_SHARED_FLOATS_PER_THREAD * reg_size : NDEGTM_SHARED_FLOATS_PER_THREAD * reg_size);
      } else {
        return 0;
      }
#else
     int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
     return ((in->TwistFlavor() == QUDA_TWIST_PLUS || in->TwistFlavor() == QUDA_TWIST_MINUS) ? DSLASH_SHARED_FLOATS_PER_THREAD * reg_size : NDEGTM_SHARED_FLOATS_PER_THREAD * reg_size);
#endif
    }

  public:
    TwistedDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
		      const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
		      const cudaColorSpinorField *x, const double kappa, const double mu, 
		      const double epsilon, const double k, const int dagger)
      : SharedDslashCuda(out, in, x),gauge0(gauge0), gauge1(gauge1), 
	reconstruct(reconstruct), dagger(dagger)
    { 
      bindSpinorTex<sFloat>(in, out, x); 

      if((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS))
      {
        setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);
        if (x) b *= epsilon; //reuse this parameter for degenerate twisted mass 
        c = 0;
      }
      else{//twist doublet:
        a = kappa, b = mu, c = epsilon, d = k;
      }
    }
    virtual ~TwistedDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
#ifdef SHARED_WILSON_DSLASH
      if (dslashParam.kernel_type == EXTERIOR_KERNEL_X) 
        errorQuda("Shared dslash does not yet support X-dimension partitioning");
#endif
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);

      if((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS)){
        DSLASH(twistedMassDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	     (sFloat*)in->V(), (float*)in->Norm(), a, b, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0));
      }
      else{
        NDEG_TM_DSLASH(twistedNdegMassDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	     (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	     (sFloat*)in->V(), (float*)in->Norm(), a, b, c, d, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0));
      }
    }

    long long flops() const { return (x ? 1416ll : 1392ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };

  template <typename sFloat, typename gFloat>
  class DomainWallDslashCuda : public DslashCuda {

  private:
    const gFloat *gauge0, *gauge1;
    const QudaReconstructType reconstruct;
    const int dagger;
    const double mferm;
    const double a;

    bool checkGrid(TuneParam &param) const {
      if (param.grid.x > deviceProp.maxGridSize[0] || param.grid.y > deviceProp.maxGridSize[1]) {
	warningQuda("Autotuner is skipping blockDim=(%u,%u,%u), gridDim=(%u,%u,%u) because lattice volume is too large",
		    param.block.x, param.block.y, param.block.z, 
		    param.grid.x, param.grid.y, param.grid.z);
	return false;
      } else {
	return true;
      }
    }

  protected:
    bool advanceBlockDim(TuneParam &param) const
    {
      const unsigned int max_shared = 16384; // FIXME: use deviceProp.sharedMemPerBlock;
      const int step[2] = { deviceProp.warpSize, 1 };
      bool advance[2] = { false, false };

      // first try to advance block.x
      param.block.x += step[0];
      if (param.block.x > deviceProp.maxThreadsDim[0] || 
	  sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
	advance[0] = false;
	param.block.x = step[0]; // reset block.x
      } else {
	advance[0] = true; // successfully advanced block.x
      }

      if (!advance[0]) {  // if failed to advance block.x, now try block.y
	param.block.y += step[1];

	if (param.block.y > in->X(4) || 
	  sharedBytesPerThread()*param.block.x*param.block.y > max_shared) {
	  advance[1] = false;
	  param.block.y = step[1]; // reset block.x
	} else {
	  advance[1] = true; // successfully advanced block.y
	}
      }

      if (advance[0] || advance[1]) {
	param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			   (in->X(4)+param.block.y-1) / param.block.y, 1);

	bool advance = true;
	if (!checkGrid(param)) advance = advanceBlockDim(param);
	return advance;
      } else {
	return false;
      }
    }

    int sharedBytesPerThread() const { return 0; }
  
  public:
    DomainWallDslashCuda(cudaColorSpinorField *out, const gFloat *gauge0, const gFloat *gauge1, 
			 const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
			 const cudaColorSpinorField *x, const double mferm, 
			 const double a, const int dagger)
      : DslashCuda(out, in, x), gauge0(gauge0), gauge1(gauge1), mferm(mferm), 
	reconstruct(reconstruct), dagger(dagger), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
    }
    virtual ~DomainWallDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    virtual void initTuneParam(TuneParam &param) const
    {
      Tunable::initTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    /** sets default values for when tuning is disabled */
    virtual void defaultTuneParam(TuneParam &param) const
    {
      Tunable::defaultTuneParam(param);
      param.grid = dim3( (dslashParam.threads+param.block.x-1) / param.block.x, 
			 (in->X(4)+param.block.y-1) / param.block.y, 1);
      bool ok = true;
      if (!checkGrid(param)) ok = advanceBlockDim(param);
      if (!ok) errorQuda("Lattice volume is too large for even the largest blockDim");
    }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream ls, recon;
      ls << dslashConstants.Ls;
      recon << reconstruct;
      key.volume += "x" + ls.str();
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Xpay";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
       if (dslashParam.kernel_type != INTERIOR_KERNEL){
         TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
         DSLASH(domainWallDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	        (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	        (sFloat*)in->V(), (float*)in->Norm(), mferm, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
       }
       else{//for debug only:
#define MAXSMSIZE 36864
         TuneParam tp;
         tp.block = dim3(dslashConstants.x[0]/2 * 4, Ls, 1);//hard-coded,und nur fuer die pruefung!
         tp.shared_bytes = tp.block.x * /*two parities*/2 * reconstruct*sizeof(gFloat);//this must be an input parameter!
         int Vh4 = dslashConstants.x[0]/2*dslashConstants.x[1]*dslashConstants.x[2]*dslashConstants.x[3];
         tp.grid = dim3( (Vh4+tp.block.x-1) / tp.block.x, (Ls + tp.block.y-1)/ tp.block.y, 1);

         DSLASH(domainWallDslash, tp.grid, tp.block, tp.shared_bytes, stream, dslashParam,
	        (sFloat*)out->V(), (float*)out->Norm(), gauge0, gauge1, 
	        (sFloat*)in->V(), (float*)in->Norm(), mferm, (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
         checkCudaError();
       }
    }

    long long flops() const { // FIXME for multi-GPU
      long long bulk = (dslashConstants.Ls-2)*(dslashConstants.VolumeCB()/dslashConstants.Ls);
      long long wall = 2*dslashConstants.VolumeCB()/dslashConstants.Ls;
      return (x ? 1368ll : 1320ll)*dslashConstants.VolumeCB()*dslashConstants.Ls + 96ll*bulk + 120ll*wall;
    }
  };

  template <typename sFloat, typename fatGFloat, typename longGFloat>
  class StaggeredDslashCuda : public DslashCuda {

  private:
    const fatGFloat *fat0, *fat1;
    const longGFloat *long0, *long1;
    const QudaReconstructType reconstruct;
    const int dagger;
    const double a;

  protected:
    int sharedBytesPerThread() const
    {
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return 6 * reg_size;
    }

  public:
    StaggeredDslashCuda(cudaColorSpinorField *out, const fatGFloat *fat0, const fatGFloat *fat1,
			const longGFloat *long0, const longGFloat *long1,
			const QudaReconstructType reconstruct, const cudaColorSpinorField *in,
			const cudaColorSpinorField *x, const double a, const int dagger)
      : DslashCuda(out, in, x), fat0(fat0), fat1(fat1), long0(long0), long1(long1),
	reconstruct(reconstruct), dagger(dagger), a(a)
    { 
      bindSpinorTex<sFloat>(in, out, x);
    }

    virtual ~StaggeredDslashCuda() { unbindSpinorTex<sFloat>(in, out, x); }

    TuneKey tuneKey() const
    {
      TuneKey key = DslashCuda::tuneKey();
      std::stringstream recon;
      recon << reconstruct;
      key.aux += ",reconstruct=" + recon.str();
      if (x) key.aux += ",Axpy";
      return key;
    }

    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      STAGGERED_DSLASH(gridDim, tp.block, tp.shared_bytes, stream, dslashParam,
		       (sFloat*)out->V(), (float*)out->Norm(), fat0, fat1, long0, long1, 
		       (sFloat*)in->V(), (float*)in->Norm(), (sFloat*)(x ? x->V() : 0), (float*)(x ? x->Norm() : 0), a);
    }

    int Nface() { return 6; }

    long long flops() const { return (x ? 1158ll : 1146ll) * dslashConstants.VolumeCB(); } // FIXME for multi-GPU
  };

#ifdef DSLASH_PROFILING

#define TDIFF(a,b) 1e3*(b.tv_sec - a.tv_sec + 1e-6*(b.tv_usec - a.tv_usec))

  void dslashTimeProfile() {

    hipEventSynchronize(dslashEnd);
    float runTime;
    hipEventElapsedTime(&runTime, dslashStart, dslashEnd);
    dslashTime += runTime;

    for (int i=4; i>=0; i--) {
      if (!dslashParam.commDim[i] && i<4) continue;

      // kernel timing
      hipEventElapsedTime(&runTime, dslashStart, kernelStart[2*i]);
      kernelTime[2*i][0] += runTime; // start time
      hipEventElapsedTime(&runTime, dslashStart, kernelEnd[2*i]);
      kernelTime[2*i][1] += runTime; // end time
    }
      
#ifdef MULTI_GPU
    for (int i=3; i>=0; i--) {
      if (!dslashParam.commDim[i]) continue;

      for (int dir = 0; dir < 2; dir ++) {
	// pack timing
	hipEventElapsedTime(&runTime, dslashStart, packStart[2*i+dir]);
	packTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, packEnd[2*i+dir]);
	packTime[2*i+dir][1] += runTime; // end time
  
	// gather timing
	hipEventElapsedTime(&runTime, dslashStart, gatherStart[2*i+dir]);
	gatherTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, gatherEnd[2*i+dir]);
	gatherTime[2*i+dir][1] += runTime; // end time
      
	// comms timing
	runTime = TDIFF(dslashStart_h, commsStart[2*i+dir]);
	commsTime[2*i+dir][0] += runTime; // start time
	runTime = TDIFF(dslashStart_h, commsEnd[2*i+dir]);
	commsTime[2*i+dir][1] += runTime; // end time

	// scatter timing
	hipEventElapsedTime(&runTime, dslashStart, scatterStart[2*i+dir]);
	scatterTime[2*i+dir][0] += runTime; // start time
	hipEventElapsedTime(&runTime, dslashStart, scatterEnd[2*i+dir]);
	scatterTime[2*i+dir][1] += runTime; // end time
      }
    }
#endif

  }

  void printDslashProfile() {
  
    printfQuda("Total Dslash time = %6.2f\n", dslashTime);

    char dimstr[8][8] = {"X-", "X+", "Y-", "Y+", "Z-", "Z+", "T-", "T+"};

    printfQuda("     %13s %13s %13s %13s %13s\n", "Pack", "Gather", "Comms", "Scatter", "Kernel");
    printfQuda("         %6s %6s %6s %6s %6s %6s %6s %6s %6s %6s\n", 
	       "Start", "End", "Start", "End", "Start", "End", "Start", "End", "Start", "End");

    printfQuda("%8s %55s %6.2f %6.2f\n", "Interior", "", kernelTime[8][0], kernelTime[8][1]);
      
    for (int i=3; i>=0; i--) {
      if (!dslashParam.commDim[i]) continue;

      for (int dir = 0; dir < 2; dir ++) {
	printfQuda("%8s ", dimstr[2*i+dir]);
#ifdef MULTI_GPU
	printfQuda("%6.2f %6.2f ", packTime[2*i+dir][0], packTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", gatherTime[2*i+dir][0], gatherTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", commsTime[2*i+dir][0], commsTime[2*i+dir][1]);
	printfQuda("%6.2f %6.2f ", scatterTime[2*i+dir][0], scatterTime[2*i+dir][1]);
#endif

	if (dir==0) printfQuda("%6.2f %6.2f\n", kernelTime[2*i][0], kernelTime[2*i][1]);
	else printfQuda("\n");
      }
    }

  }
#endif

  int gatherCompleted[Nstream];
  int previousDir[Nstream];
  int commsCompleted[Nstream];
  int dslashCompleted[Nstream];
  int commDimTotal;

  /**
   * Initialize the arrays used for the dynamic scheduling.
   */
  void initDslashCommsPattern() {
    for (int i=0; i<Nstream-1; i++) {
      gatherCompleted[i] = 0;
      commsCompleted[i] = 0;
      dslashCompleted[i] = 0;
    }
    gatherCompleted[Nstream-1] = 1;
    commsCompleted[Nstream-1] = 1;

    //   We need to know which was the previous direction in which
    //   communication was issued, since we only query a given event /
    //   comms call after the previous the one has successfully
    //   completed.
    for (int i=3; i>=0; i--) {
      if (dslashParam.commDim[i]) {
	int prev = Nstream-1;
	for (int j=3; j>i; j--) if (dslashParam.commDim[j]) prev = 2*j;
	previousDir[2*i + 1] = prev;
	previousDir[2*i + 0] = 2*i + 1; // always valid
      }
    }

    // this tells us how many events / comms occurances there are in
    // total.  Used for exiting the while loop
    commDimTotal = 0;
    for (int i=3; i>=0; i--) commDimTotal += dslashParam.commDim[i];
    commDimTotal *= 4; // 2 from pipe length, 2 from direction
  }

  void dslashCuda(DslashCuda &dslash, const size_t regSize, const int parity, const int dagger, 
		  const int volume, const int *faceVolumeCB) {

    dslashParam.parity = parity;
    dslashParam.kernel_type = INTERIOR_KERNEL;
    dslashParam.threads = volume;

    gettimeofday(&dslashStart_h, NULL);

#ifdef MULTI_GPU
    // Record the start of the dslash
    hipEventRecord(dslashStart, streams[Nstream-1]);
    // Initialize pack from source spinor
    face->pack(*inSpinor, 1-parity, dagger, streams);
    
    // Record the end of the packing
    hipEventRecord(packEnd[0], streams[Nstream-1]);

    for(int i = 3; i >=0; i--){
      if (!dslashParam.commDim[i]) continue;

      for (int dir=1; dir>=0; dir--) {
        if (i!=3 || getKernelPackT())
          hipStreamWaitEvent(streams[2*i+dir], packEnd[0], 0);
        else
          hipStreamWaitEvent(streams[2*i+dir], dslashStart, 0);

	// Initialize host transfer from source spinor
	face->gather(*inSpinor, dagger, 2*i+dir);

	// Record the end of the gathering
	hipEventRecord(gatherEnd[2*i+dir], streams[2*i+dir]);
      }
    }
#endif

    dslash.apply(streams[Nstream-1]);

#ifdef MULTI_GPU
    initDslashCommsPattern();

    int completeSum = 0;
    while (completeSum < commDimTotal) {
      for (int i=3; i>=0; i--) {
	if (!dslashParam.commDim[i]) continue;
      
	for (int dir=1; dir>=0; dir--) {
	
	  // Query if gather has completed
	  if (!gatherCompleted[2*i+dir] && gatherCompleted[previousDir[2*i+dir]]) { 
	    if (hipSuccess == hipEventQuery(gatherEnd[2*i+dir])) {
	      gatherCompleted[2*i+dir] = 1;
	      completeSum++;
	      gettimeofday(&commsStart[2*i+dir], NULL);
	      face->commsStart(2*i+dir);
	    }
	  }
	
	  // Query if comms has finished
	  if (!commsCompleted[2*i+dir] && commsCompleted[previousDir[2*i+dir]] &&
	      gatherCompleted[2*i+dir]) {
	    if (face->commsQuery(2*i+dir)) { 
	      commsCompleted[2*i+dir] = 1;
	      completeSum++;
	      gettimeofday(&commsEnd[2*i+dir], NULL);
	    
	      // Scatter into the end zone
	      face->scatter(*inSpinor, dagger, 2*i+dir);	    
	    }
	  }

	}
	 
	// enqueue the boundary dslash kernel as soon as the scatters have been enqueued
	if (!dslashCompleted[2*i] && commsCompleted[2*i] && commsCompleted[2*i+1] ) {
	  // Record the end of the scattering
	  hipEventRecord(scatterEnd[2*i], streams[2*i]);

	  dslashParam.kernel_type = static_cast<KernelType>(i);
	  dslashParam.threads = dslash.Nface()*faceVolumeCB[i]; // updating 2 or 6 faces
	  
	  // wait for scattering to finish and then launch dslash
	  hipStreamWaitEvent(streams[Nstream-1], scatterEnd[2*i], 0);
	  
	  dslash.apply(streams[Nstream-1]); // all faces use this stream

	  dslashCompleted[2*i] = 1;
	}

      }
    
    }
    //hipEventRecord(dslashEnd, streams[Nstream-1]);
    //DSLASH_TIME_PROFILE();
#endif // MULTI_GPU
  }

  // Wilson wrappers
  void wilsonDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const cudaColorSpinorField *in, const int parity,
			const int dagger, const cudaColorSpinorField *x, const double &k, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge %d and spinor %d precision not supported", 
		gauge.Precision(), in->Precision());

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);
    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new WilsonDslashCuda<double2, double2>(out, (double2*)gauge0, (double2*)gauge1, 
						      gauge.Reconstruct(), in, x, k, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new WilsonDslashCuda<float4, float4>(out, (float4*)gauge0, (float4*)gauge1,
						    gauge.Reconstruct(), in, x, k, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new WilsonDslashCuda<short4, short4>(out, (short4*)gauge0, (short4*)gauge1,
						    gauge.Reconstruct(), in, x, k, dagger);
    }
    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

  }

  void cloverDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover cloverInv,
			const cudaColorSpinorField *in, const int parity, const int dagger, 
			const cudaColorSpinorField *x, const double &a, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_CLOVER_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *cloverP, *cloverNormP;
    QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    if (in->Precision() != clover_prec)
      errorQuda("Mixing clover and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new CloverDslashCuda<double2, double2, double2>(out, (double2*)gauge0, (double2*)gauge1, 
							       gauge.Reconstruct(), (double2*)cloverP, 
							       (float*)cloverNormP, in, x, a, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new CloverDslashCuda<float4, float4, float4>(out, (float4*)gauge0, (float4*)gauge1,
							    gauge.Reconstruct(), (float4*)cloverP,
							    (float*)cloverNormP, in, x, a, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new CloverDslashCuda<short4, short4, short4>(out, (short4*)gauge0, (short4*)gauge1,
							    gauge.Reconstruct(), (short4*)cloverP,
							    (float*)cloverNormP, in, x, a, dagger);
    }

    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(gauge);
    unbindCloverTex(cloverInv);

    checkCudaError();
#else
    errorQuda("Clover dslash has not been built");
#endif

  }


  void asymCloverDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover cloverInv,
			    const cudaColorSpinorField *in, const int parity, const int dagger, 
			    const cudaColorSpinorField *x, const double &a, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_CLOVER_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }

    void *cloverP, *cloverNormP;
    QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    if (in->Precision() != clover_prec)
      errorQuda("Mixing clover and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new AsymCloverDslashCuda<double2, double2, double2>(out, (double2*)gauge0, (double2*)gauge1, 
								   gauge.Reconstruct(), (double2*)cloverP, 
								   (float*)cloverNormP, in, x, a, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new AsymCloverDslashCuda<float4, float4, float4>(out, (float4*)gauge0, (float4*)gauge1, 
								gauge.Reconstruct(), (float4*)cloverP, 
								(float*)cloverNormP, in, x, a, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new AsymCloverDslashCuda<short4, short4, short4>(out, (short4*)gauge0, (short4*)gauge1, 
								gauge.Reconstruct(), (short4*)cloverP, 
								(float*)cloverNormP, in, x, a, dagger);
    }

    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(gauge);
    unbindCloverTex(cloverInv);

    checkCudaError();
#else
    errorQuda("Clover dslash has not been built");
#endif

  }

  void twistedMassDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			   const cudaColorSpinorField *in, const int parity, const int dagger, 
			   const cudaColorSpinorField *x, const double &kappa, const double &mu, 
			   const double &epsilon, const double &k,  const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL
  #ifdef GPU_TWISTED_MASS_DIRAC
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
  
    int ghost_threads[4] = {0};
    int bulk_threads = ((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS)) ? in->Volume() : in->Volume() / 2;
  
    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
      ghost_threads[i] = ((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS)) ? in->GhostFace()[i] : in->GhostFace()[i] / 2;
    }

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
        errorQuda("Mixing gauge and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new TwistedDslashCuda<double2,double2>(out, (double2*)gauge0,(double2*)gauge1, gauge.Reconstruct(), in, x, kappa, mu, epsilon, k, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new TwistedDslashCuda<float4,float4>(out, (float4*)gauge0,(float4*)gauge1, gauge.Reconstruct(), in, x, kappa, mu, epsilon, k, dagger);

    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new TwistedDslashCuda<short4,short4>(out, (short4*)gauge0,(short4*)gauge1, gauge.Reconstruct(), in, x, kappa, mu, epsilon, k, dagger);
    }

    dslashCuda(*dslash, regSize, parity, dagger, bulk_threads, ghost_threads);

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Twisted mass dslash has not been built");
#endif
  }

  void domainWallDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			    const cudaColorSpinorField *in, const int parity, const int dagger, 
			    const cudaColorSpinorField *x, const double &m_f, const double &k2, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

    dslashParam.parity = parity;

#ifdef GPU_DOMAIN_WALL_DIRAC
    //currently splitting in space-time is impelemented:
    int dirs = 4;
    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
    for(int i = 0;i < dirs; i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }  

    void *gauge0, *gauge1;
    bindGaugeTex(gauge, parity, &gauge0, &gauge1);

    if (in->Precision() != gauge.Precision())
      errorQuda("Mixing gauge and spinor precision not supported");

    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new DomainWallDslashCuda<double2,double2>(out, (double2*)gauge0, (double2*)gauge1, 
							 gauge.Reconstruct(), in, x, m_f, k2, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new DomainWallDslashCuda<float4,float4>(out, (float4*)gauge0, (float4*)gauge1, 
						       gauge.Reconstruct(), in, x, m_f, k2, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      dslash = new DomainWallDslashCuda<short4,short4>(out, (short4*)gauge0, (short4*)gauge1, 
						       gauge.Reconstruct(), in, x, m_f, k2, dagger);
    }

    // the parameters passed to dslashCuda must be 4-d volume and 3-d
    // faces because Ls is added as the y-dimension in thread space
    int ghostFace[QUDA_MAX_DIM];
    for (int i=0; i<4; i++) ghostFace[i] = in->GhostFace()[i] / in->X(4);
    dslashCuda(*dslash, regSize, parity, dagger, in->Volume() / in->X(4), ghostFace);

    delete dslash;
    unbindGaugeTex(gauge);

    checkCudaError();
#else
    errorQuda("Domain wall dslash has not been built");
#endif
  }

  void staggeredDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &fatGauge, 
			   const cudaGaugeField &longGauge, const cudaColorSpinorField *in,
			   const int parity, const int dagger, const cudaColorSpinorField *x,
			   const double &k, const int *commOverride)
  {
    inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

#ifdef MULTI_GPU
    for(int i=0;i < 4; i++){
      if(commDimPartitioned(i) && (fatGauge.X()[i] < 6)){
	errorQuda("ERROR: partitioned dimension with local size less than 6 is not supported in staggered dslash\n");
      }    
    }
#endif

    int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code

    dslashParam.parity = parity;

    for(int i=0;i<4;i++){
      dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
      dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
      dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
      dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
    }
    void *fatGauge0, *fatGauge1;
    void* longGauge0, *longGauge1;
    bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
    bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    
    if (in->Precision() != fatGauge.Precision() || in->Precision() != longGauge.Precision()){
      errorQuda("Mixing gauge and spinor precision not supported"
		"(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
		in->Precision(), fatGauge.Precision(), longGauge.Precision());
    }
    
    DslashCuda *dslash = 0;
    size_t regSize = sizeof(float);

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      dslash = new StaggeredDslashCuda<double2, double2, double2>(out, (double2*)fatGauge0, (double2*)fatGauge1,
								  (double2*)longGauge0, (double2*)longGauge1, 
								  longGauge.Reconstruct(), in, x, k, dagger);
      regSize = sizeof(double);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      dslash = new StaggeredDslashCuda<float2, float2, float4>(out, (float2*)fatGauge0, (float2*)fatGauge1,
							       (float4*)longGauge0, (float4*)longGauge1, 
							       longGauge.Reconstruct(), in, x, k, dagger);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {	
      dslash = new StaggeredDslashCuda<short2, short2, short4>(out, (short2*)fatGauge0, (short2*)fatGauge1,
							       (short4*)longGauge0, (short4*)longGauge1, 
							       longGauge.Reconstruct(), in, x, k, dagger);
    }

    dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), in->GhostFace());

    delete dslash;
    unbindGaugeTex(fatGauge);
    unbindGaugeTex(longGauge);

    checkCudaError();
  
#else
    errorQuda("Staggered dslash has not been built");
#endif  // GPU_STAGGERED_DIRAC
  }


  template <typename sFloat, typename cFloat>
  class CloverCuda : public Tunable {
  private:
    cudaColorSpinorField *out;
    float *outNorm;
    char *saveOut, *saveOutNorm;
    const cFloat *clover;
    const float *cloverNorm;
    const cudaColorSpinorField *in;

  protected:
    int sharedBytesPerThread() const
    {
      int reg_size = (typeid(sFloat)==typeid(double2) ? sizeof(double) : sizeof(float));
      return CLOVER_SHARED_FLOATS_PER_THREAD * reg_size;
    }
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool advanceGridDim(TuneParam &param) const { return false; } // Don't tune the grid dimensions.

  public:
    CloverCuda(cudaColorSpinorField *out, const cFloat *clover, const float *cloverNorm, 
	       const cudaColorSpinorField *in)
      : out(out), clover(clover), cloverNorm(cloverNorm), in(in)
    {
      bindSpinorTex<sFloat>(in);
    }
    virtual ~CloverCuda() { unbindSpinorTex<sFloat>(in); }
    void apply(const hipStream_t &stream)
    {
      TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
      dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
      cloverKernel<<<gridDim, tp.block, tp.shared_bytes, stream>>>
	((sFloat*)out->V(), (float*)out->Norm(), clover, cloverNorm, 
	 (sFloat*)in->V(), (float*)in->Norm(), dslashParam);
    }
    virtual TuneKey tuneKey() const
    {
      std::stringstream vol, aux;
      vol << dslashConstants.x[0] << "x";
      vol << dslashConstants.x[1] << "x";
      vol << dslashConstants.x[2] << "x";
      vol << dslashConstants.x[3];
      return TuneKey(vol.str(), typeid(*this).name());
    }

    // Need to save the out field if it aliases the in field
    void preTune() {
      if (in == out) {
	saveOut = new char[out->Bytes()];
	hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);
	if (typeid(sFloat) == typeid(short4)) {
	  saveOutNorm = new char[out->NormBytes()];
	  hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
	}
      }
    }

    // Restore if the in and out fields alias
    void postTune() {
      if (in == out) {
	hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
	delete[] saveOut;
	if (typeid(sFloat) == typeid(short4)) {
	  hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
	  delete[] saveOutNorm;
	}
      }
    }

    std::string paramString(const TuneParam &param) const // Don't bother printing the grid dim.
    {
      std::stringstream ps;
      ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
      ps << "shared=" << param.shared_bytes;
      return ps.str();
    }

    long long flops() const { return 504ll * dslashConstants.VolumeCB(); }
  };


  void cloverCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover clover, 
		  const cudaColorSpinorField *in, const int parity) {

    dslashParam.parity = parity;
    dslashParam.threads = in->Volume();

#ifdef GPU_CLOVER_DIRAC
    Tunable *clov = 0;
    void *cloverP, *cloverNormP;
    QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

    if (in->Precision() != clover_prec)
      errorQuda("Mixing clover and spinor precision not supported");

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      clov = new CloverCuda<double2, double2>(out, (double2*)cloverP, (float*)cloverNormP, in);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      clov = new CloverCuda<float4, float4>(out, (float4*)cloverP, (float*)cloverNormP, in);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      clov = new CloverCuda<short4, short4>(out, (short4*)cloverP, (float*)cloverNormP, in);
    }
    clov->apply(0);

    unbindCloverTex(clover);
    checkCudaError();

    delete clov;
#else
    errorQuda("Clover dslash has not been built");
#endif
  }


  template <typename sFloat>
  class TwistGamma5Cuda : public Tunable {

  private:
    cudaColorSpinorField *out;
    const cudaColorSpinorField *in;
    double a;
    double b;
    double c;

    int sharedBytesPerThread() const { return 0; }
    int sharedBytesPerBlock(const TuneParam &param) const { return 0; }
    bool advanceGridDim(TuneParam &param) const { return false; } // Don't tune the grid dimensions.

    char *saveOut, *saveOutNorm;

  public:
    TwistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		    double kappa, double mu, double epsilon, const int dagger, QudaTwistGamma5Type twist) :
      out(out), in(in) 
    {
      bindSpinorTex<sFloat>(in);
      if((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS))
        setTwistParam(a, b, kappa, mu, dagger, twist);
      else{//twist doublet
        a = kappa, b = mu, c = epsilon;
      } 
    }
    virtual ~TwistGamma5Cuda() {
      unbindSpinorTex<sFloat>(in);    
    }

   TuneKey tuneKey() const {
     std::stringstream vol, aux;
     vol << dslashConstants.x[0] << "x";
     vol << dslashConstants.x[1] << "x";
     vol << dslashConstants.x[2] << "x";
     vol << dslashConstants.x[3];    
     return TuneKey(vol.str(), typeid(*this).name(), aux.str());
   }  

  void apply(const hipStream_t &stream) 
  {
    TuneParam tp = tuneLaunch(*this, dslashTuning, verbosity);
    dim3 gridDim( (dslashParam.threads+tp.block.x-1) / tp.block.x, 1, 1);
    if((in->TwistFlavor() == QUDA_TWIST_PLUS) || (in->TwistFlavor() == QUDA_TWIST_MINUS))
    {
        twistGamma5Kernel<<<gridDim, tp.block, tp.shared_bytes, stream>>> 
	((sFloat*)out->V(), (float*)out->Norm(), a, b, (sFloat*)in->V(), (float*)in->Norm(), dslashParam);
    }
    else
    {
        twistGamma5Kernel<<<gridDim, tp.block, tp.shared_bytes, stream>>>
	((sFloat*)out->V(), (float*)out->Norm(), a, b, c, (sFloat*)in->V(), (float*)in->Norm(), dslashParam);
    }
  }

  void preTune() {
    saveOut = new char[out->Bytes()];
    hipMemcpy(saveOut, out->V(), out->Bytes(), hipMemcpyDeviceToHost);
    if (typeid(sFloat) == typeid(short4)) {
      saveOutNorm = new char[out->NormBytes()];
      hipMemcpy(saveOutNorm, out->Norm(), out->NormBytes(), hipMemcpyDeviceToHost);
    }
  }

  void postTune() {
    hipMemcpy(out->V(), saveOut, out->Bytes(), hipMemcpyHostToDevice);
    delete[] saveOut;
    if (typeid(sFloat) == typeid(short4)) {
      hipMemcpy(out->Norm(), saveOutNorm, out->NormBytes(), hipMemcpyHostToDevice);
      delete[] saveOutNorm;
    }
  }

  std::string paramString(const TuneParam &param) const {
    std::stringstream ps;
    ps << "block=(" << param.block.x << "," << param.block.y << "," << param.block.z << "), ";
    ps << "shared=" << param.shared_bytes;
    return ps.str();
  }

  long long flops() const { return 24ll * dslashConstants.VolumeCB(); }
 };

//!ndeg tm: 
  void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const int dagger, const double &kappa, const double &mu, const double &epsilon,   const QudaTwistGamma5Type twist)
  {
    if(in->TwistFlavor() == QUDA_TWIST_PLUS || in->TwistFlavor() == QUDA_TWIST_MINUS)
      dslashParam.threads = in->Volume();
    else //twist doublet    
      dslashParam.threads = in->Volume() / 2;

#if (defined GPU_TWISTED_MASS_DIRAC) || (defined GPU_NDEG_TWISTED_MASS_DIRAC)
    Tunable *twistGamma5 = 0;

    if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__COMPUTE_CAPABILITY__ >= 130)
      twistGamma5 = new TwistGamma5Cuda<double2>(out, in, kappa, mu, epsilon, dagger, twist);
#else
      errorQuda("Double precision not supported on this GPU");
#endif
    } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
      twistGamma5 = new TwistGamma5Cuda<float4>(out, in, kappa, mu, epsilon, dagger, twist);
    } else if (in->Precision() == QUDA_HALF_PRECISION) {
      twistGamma5 = new TwistGamma5Cuda<short4>(out, in, kappa, mu, epsilon, dagger, twist);
    }

    twistGamma5->apply(streams[Nstream-1]);
    checkCudaError();

    delete twistGamma5;
#else
    errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
  }

} // namespace quda

#include "misc_helpers.cu"


#if defined(GPU_FATLINK) || defined(GPU_GAUGE_FORCE) || defined(GPU_FERMION_FORCE) || defined(GPU_HISQ_FORCE) || defined(GPU_UNITARIZE)
#include <force_common.h>
#endif

#ifdef GPU_FATLINK
#include "llfat_quda.cu"
#endif

#ifdef GPU_GAUGE_FORCE
#include "gauge_force_quda.cu"
#endif

#ifdef GPU_FERMION_FORCE
#include "fermion_force_quda.cu"
#endif

#ifdef GPU_UNITARIZE
#include "unitarize_links_quda.cu"
#endif

#ifdef GPU_HISQ_FORCE
#include "hisq_paths_force_quda.cu"
#include "unitarize_force_quda.cu"
#endif

