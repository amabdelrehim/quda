#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

#include <color_spinor_field.h>
#include <clover_field.h>

#define BLOCK_DIM 64

// these control the Wilson-type actions
//#define DIRECT_ACCESS_LINK
//#define DIRECT_ACCESS_WILSON_SPINOR
//#define DIRECT_ACCESS_WILSON_ACCUM
//#define DIRECT_ACCESS_WILSON_INTER
//#define DIRECT_ACCESS_WILSON_PACK_SPINOR

//these are access control for staggered action
#if (__CUDA_ARCH__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
#endif

#include <quda_internal.h>
#include <dslash_quda.h>
#include <sys/time.h>

//#define PARALLEL_DIR

enum KernelType {
  INTERIOR_KERNEL = 5,
  EXTERIOR_KERNEL_X = 0,
  EXTERIOR_KERNEL_Y = 1,
  EXTERIOR_KERNEL_Z = 2,
  EXTERIOR_KERNEL_T = 3
};

struct DslashParam {
  int tOffset; // offset into the T dimension (multi gpu only)
  int tMul;    // spatial volume distance between the T faces being updated (multi gpu only)
  int threads; // the desired number of active threads
  int parity;  // Even-Odd or Odd-Even
  int commDim[QUDA_MAX_DIM]; // Whether to do comms or not
  int ghostDim[QUDA_MAX_DIM]; // Whether a ghost zone has been allocated for a given dimension
  int ghostOffset[QUDA_MAX_DIM];
  int ghostNormOffset[QUDA_MAX_DIM];
  KernelType kernel_type; //is it INTERIOR_KERNEL, EXTERIOR_KERNEL_X/Y/Z/T
};

// determines whether the temporal ghost zones are packed with a gather kernel,
// as opposed to multiple calls to hipMemcpy()
bool kernelPackT = false;

DslashParam dslashParam;

// these are set in initDslashConst
int Vspatial;
#ifdef MULTI_GPU
static const int Nstream = 9;
#else
static const int Nstream = 1;
#endif
static hipStream_t streams[Nstream];
static hipEvent_t scatterEvent[Nstream];
static hipEvent_t dslashEnd;

FaceBuffer *face;
cudaColorSpinorField *inSpinor;

#include <dslash_textures.h>
#include <dslash_constants.h>

#define SHORT_LENGTH 65536
#define SCALE_FLOAT ((SHORT_LENGTH-1) * 0.5) // 32767.5
#define SHIFT_FLOAT (-1.f / (SHORT_LENGTH-1)) // 1.5259021897e-5

#if defined(DIRECT_ACCESS_LINK) || defined(DIRECT_ACCESS_WILSON_SPINOR) || \
  defined(DIRECT_ACCESS_WILSON_ACCUM) || defined(DIRECT_ACCESS_WILSON_PACK_SPINOR)
static inline __device__ short float2short(float c, float a) {
  //return (short)(a*MAX_SHORT);
  short rtn = (short)((a+SHIFT_FLOAT)*SCALE_FLOAT*c);
  return rtn;
}

static inline __device__ float short2float(short a) {
  return (float)a/SCALE_FLOAT - SHIFT_FLOAT;
}

static inline __device__ short4 float42short4(float c, float4 a) {
  return make_short4(float2short(c, a.x), float2short(c, a.y), float2short(c, a.z), float2short(c, a.w));
}

static inline __device__ float4 short42float4(short4 a) {
  return make_float4(short2float(a.x), short2float(a.y), short2float(a.z), short2float(a.w));
}

static inline __device__ float2 short22float2(short2 a) {
  return make_float2(short2float(a.x), short2float(a.y));
}
#endif // DIRECT_ACCESS inclusions

#include <inline_ptx.h>

// dslashTuning = QUDA_TUNE_YES turns off error checking
static QudaTune dslashTuning = QUDA_TUNE_NO;

void setDslashTuning(QudaTune tune)
{
  dslashTuning = tune;
}

#if (CUDA_VERSION <= 4000)
#define VOLATILE volatile
#else
#define VOLATILE 
#endif

#include <pack_face_def.h>        // kernels for packing the ghost zones and general indexing
#include <staggered_dslash_def.h> // staggered Dslash kernels
#include <wilson_dslash_def.h>    // Wilson Dslash kernels (including clover)
#include <dw_dslash_def.h>        // Domain Wall kernels
#include <tm_dslash_def.h>        // Twisted Mass kernels
#include <tm_core.h>              // solo twisted mass kernel
#include <clover_def.h>           // kernels for applying the clover term alone

#undef VOLATILE

#ifndef DSLASH_SHARED_FLOATS_PER_THREAD
#define DSLASH_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef CLOVER_SHARED_FLOATS_PER_THREAD
#define CLOVER_SHARED_FLOATS_PER_THREAD 0
#endif

#ifndef SHARED_COORDS
#define SHARED_COORDS 0
#endif

#include <blas_quda.h>
#include <face_quda.h>


__global__ void dummyKernel() {
  // do nothing
}

void initCache() {

#if (__CUDA_ARCH__ >= 200)

  static int firsttime = 1;
  if (firsttime){	
    hipFuncSetCacheConfig(reinterpret_cast<const void*>(dummyKernel), hipFuncCachePreferL1);
    dummyKernel<<<1,1>>>();
    firsttime=0;
  }

#endif

}

void setFace(const FaceBuffer &Face) {
  face = (FaceBuffer*)&Face; // nasty
}

#define MORE_GENERIC_DSLASH(FUNC, DAG, X, kernel_type, gridDim, blockDim, shared, stream, param,  ...)            \
  if (x==0) {                                                                                                     \
    if (reconstruct == QUDA_RECONSTRUCT_NO) {                                                                     \
      FUNC ## 18 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);     \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {                                                              \
      FUNC ## 12 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__ , param);     \
    } else {                                                                                                      \
      FUNC ## 8 ## DAG ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param);       \
    }                                                                                                             \
  } else {                                                                                                        \
    if (reconstruct == QUDA_RECONSTRUCT_NO) {                                                                     \
      FUNC ## 18 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_12) {                                                              \
      FUNC ## 12 ## DAG ## X ## Kernel<kernel_type><<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {                                                               \
      FUNC ## 8 ## DAG ## X ## Kernel<kernel_type> <<<gridDim, blockDim, shared, stream>>> ( __VA_ARGS__, param); \
    }                                                                                                             \
  }

#ifndef MULTI_GPU

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...)                          \
  switch(param.kernel_type) {						                                      \
  case INTERIOR_KERNEL:							                                      \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                      \
  default:								                                      \
    errorQuda("KernelType %d not defined for single GPU", param.kernel_type);                                 \
  }

#else

#define GENERIC_DSLASH(FUNC, DAG, X, gridDim, blockDim, shared, stream, param,  ...)                            \
  switch(param.kernel_type) {						                                        \
  case INTERIOR_KERNEL:							                                        \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, INTERIOR_KERNEL,   gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                        \
  case EXTERIOR_KERNEL_X:							                                \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_X, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                        \
  case EXTERIOR_KERNEL_Y:							                                \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Y, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                        \
  case EXTERIOR_KERNEL_Z:							                                \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_Z, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                        \
  case EXTERIOR_KERNEL_T:							                                \
    MORE_GENERIC_DSLASH(FUNC, DAG, X, EXTERIOR_KERNEL_T, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
    break;								                                        \
  }

#endif

// macro used for dslash types with dagger kernel defined (Wilson, domain wall, etc.)
#define DSLASH(FUNC, gridDim, blockDim, shared, stream, param, ...)	\
  if (!dagger) {							\
    GENERIC_DSLASH(FUNC, , Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
  } else {								\
    GENERIC_DSLASH(FUNC, Dagger, Xpay, gridDim, blockDim, shared, stream, param, __VA_ARGS__) \
 }

// macro used for staggered dslash
#define STAGGERED_DSLASH(gridDim, blockDim, shared, stream, param, ...)	\
    GENERIC_DSLASH(staggeredDslash, , Axpy, gridDim, blockDim, shared, stream, param, __VA_ARGS__)


// Use an abstract class interface to drive the different CUDA dslash
// kernels.  All parameters are curried into the derived classes to
// allow a simple interface.
class DslashCuda {
public:
  DslashCuda() { ; }
  virtual ~DslashCuda() { ; }
  virtual void apply(const dim3 &blockDim, const dim3 &gridDim, const int shared_bytes, const hipStream_t &stream) = 0;
};


template <typename sFloat, typename gFloat>
class WilsonDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double a;

public:
  WilsonDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), in(in), 
    inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }

  virtual ~WilsonDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const dim3 &gridDim, const int shared_bytes, const hipStream_t &stream) {
    //dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    //printfQuda("Applying dslash: threads = %d, type = %d\n", dslashParam.threads, dslashParam.kernel_type);
    DSLASH(dslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, in, inNorm, x, xNorm, a);
  }

};

template <typename sFloat, typename gFloat, typename cFloat>
class CloverDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const cFloat *clover;
  const float *cloverNorm;
  const int dagger;
  const double a;

public:
  CloverDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		   const QudaReconstructType reconstruct, const cFloat *clover, 
		   const float *cloverNorm, const sFloat *in, const float *inNorm,
		   const sFloat *x, const float *xNorm, const double a,
		   const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    clover(clover), cloverNorm(cloverNorm), in(in), inNorm(inNorm), 
    reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }
  virtual ~CloverDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const dim3 &gridDim, const int shared_bytes, const hipStream_t &stream) {
    //dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(cloverDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, x, xNorm, a);
  }

};

void setTwistParam(double &a, double &b, const double &kappa, const double &mu, 
		   const int dagger, const QudaTwistGamma5Type twist) {
  if (twist == QUDA_TWIST_GAMMA5_DIRECT) {
    a = 2.0 * kappa * mu;
    b = 1.0;
  } else if (twist == QUDA_TWIST_GAMMA5_INVERSE) {
    a = -2.0 * kappa * mu;
    b = 1.0 / (1.0 + a*a);
  } else {
    errorQuda("Twist type %d not defined\n", twist);
  }
  if (dagger) a *= -1.0;

}

template <typename sFloat, typename gFloat>
class TwistedDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  double a;
  double b;

public:
  TwistedDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		    const QudaReconstructType reconstruct, const sFloat *in, const float *inNorm,
		    const sFloat *x, const float *xNorm, const double kappa, const double mu,
		    const double k, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
    setTwistParam(a, b, kappa, mu, dagger, QUDA_TWIST_GAMMA5_INVERSE);
    if (x) b *= k;
  }
  virtual ~TwistedDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const dim3 &gridDim, const int shared_bytes, const hipStream_t &stream) {
    //dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(twistedMassDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
	   out, outNorm, gauge0, gauge1, in, inNorm, a, b, x, xNorm);
  }

};

template <typename sFloat, typename gFloat>
class DomainWallDslashCuda : public DslashCuda {

private:
  sFloat *out;
  float *outNorm;
  const sFloat *in, *x;
  const float *inNorm, *xNorm;
  const gFloat *gauge0, *gauge1;
  const QudaReconstructType reconstruct;
  const int dagger;
  const double mferm;
  const double a;

public:
  DomainWallDslashCuda(sFloat *out, float *outNorm, const gFloat *gauge0, const gFloat *gauge1, 
		       const QudaReconstructType reconstruct, const sFloat *in, 
		       const float *inNorm, const sFloat *x, const float *xNorm, const double mferm, 
		       const double a, const int dagger, const size_t bytes, const size_t norm_bytes) :
    DslashCuda(), out(out), outNorm(outNorm), gauge0(gauge0), gauge1(gauge1), 
    in(in), inNorm(inNorm), mferm(mferm), reconstruct(reconstruct), dagger(dagger), x(x), xNorm(xNorm), a(a) { 
    bindSpinorTex(bytes, norm_bytes, in, inNorm, out, outNorm, x, xNorm); 
  }
  virtual ~DomainWallDslashCuda() { unbindSpinorTex(in, inNorm, out, outNorm, x, xNorm); }

  void apply(const dim3 &blockDim, const dim3 &gridDim, const int shared_bytes, const hipStream_t &stream) {
    //dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);
    DSLASH(domainWallDslash, gridDim, blockDim, shared_bytes, stream, dslashParam,
    	   out, outNorm, gauge0, gauge1, in, inNorm, mferm, x, xNorm, a);
  }

};

void dslashCuda(DslashCuda &dslash, const size_t regSize, const int parity, const int dagger, 
		const int volume, const int *faceVolumeCB, const dim3 *blockDim, const dim3 *gridDim) {

  dslashParam.parity = parity;

  dslashParam.kernel_type = INTERIOR_KERNEL;
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = volume;

#ifdef MULTI_GPU
  // wait for any previous outstanding dslashes to finish
  hipStreamWaitEvent(0, dslashEnd, 0);

  // Gather from source spinor
  for(int dir = 3; dir >=0; dir--){ // count down for Wilson
    if (!dslashParam.commDim[dir]) continue;
    face->exchangeFacesStart(*inSpinor, 1-parity, dagger, dir, streams);
  }
#endif

  
  int block_xy = blockDim[0].x*blockDim[0].y;
  if (block_xy % 32 != 0) block_xy = ((block_xy / 32) + 1)*32;
  int block = block_xy*blockDim[0].z;
    
  int shared_bytes = block*(DSLASH_SHARED_FLOATS_PER_THREAD*regSize + SHARED_COORDS);
  dslash.apply(blockDim[0], gridDim[0], shared_bytes, streams[Nstream-1]); // stream 0 or 8

#ifdef MULTI_GPU

  for (int i=3; i>=0; i--) { // count down for Wilson
    if (!dslashParam.commDim[i]) continue;

    // Finish gather and start comms
    face->exchangeFacesComms(i);
    
    // Wait for comms to finish, and scatter into the end zone
    face->exchangeFacesWait(*inSpinor, dagger, i);

    // Record the end of the scattering
    hipEventRecord(scatterEvent[2*i], streams[2*i]);
    hipEventRecord(scatterEvent[2*i+1], streams[2*i+1]);
  }

  for (int i=3; i>=0; i--) { // count down for Wilson
    if (!dslashParam.commDim[i]) continue;

    shared_bytes = blockDim[i+1].x*(DSLASH_SHARED_FLOATS_PER_THREAD*regSize + SHARED_COORDS);
    
    //hipStreamSynchronize(streams[2*i]);
    //hipStreamSynchronize(streams[2*i + 1]);
    
    dslashParam.kernel_type = static_cast<KernelType>(i);
    //dslashParam.tOffset = dims[i]-2; // is this redundant?
    dslashParam.threads = 2*faceVolumeCB[i]; // updating 2 faces

    // wait for scattering to finish and then launch dslash
    hipStreamWaitEvent(streams[Nstream-1], scatterEvent[2*i], 0);
    hipStreamWaitEvent(streams[Nstream-1], scatterEvent[2*i+1], 0);
    dslash.apply(blockDim[i+1], gridDim[i+1], shared_bytes, streams[Nstream-1]); // all faces use this stream
  }

  hipEventRecord(dslashEnd, streams[Nstream-1]);
  //hipStreamSynchronize(streams[Nstream-1]);

#endif // MULTI_GPU
}

// Wilson wrappers
void wilsonDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const cudaColorSpinorField *in,
		      const int parity, const int dagger, const cudaColorSpinorField *x,
		      const double &k, const dim3 *blockDim, const dim3 *gridDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_WILSON_DIRAC
  int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
    dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->Precision() != gauge.Precision())
    errorQuda("Mixing gauge and spinor precision not supported");

  const void *xv = (x ? x->V() : 0);
  const void *xn = (x ? x->Norm() : 0);

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);
  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new WilsonDslashCuda<double2, double2>((double2*)out->V(), (float*)out->Norm(), (double2*)gauge0, (double2*)gauge1, 
						    gauge.Reconstruct(), (double2*)in->V(), (float*)in->Norm(), 
						    (double2*)xv, (float*)xn, k, dagger, in->Bytes(), in->NormBytes());
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    dslash = new WilsonDslashCuda<float4, float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, (float4*)gauge1,
						  gauge.Reconstruct(), (float4*)in->V(), (float*)in->Norm(), 
						  (float4*)xv, (float*)xn, k, dagger, in->Bytes(), in->NormBytes());
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    dslash = new WilsonDslashCuda<short4, short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, (short4*)gauge1,
						  gauge.Reconstruct(), (short4*)in->V(), (float*)in->Norm(),
						  (short4*)xv, (float*)xn, k, dagger, in->Bytes(), in->NormBytes());
  }
  dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), 
	     in->GhostFace(), blockDim, gridDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Wilson dslash has not been built");
#endif // GPU_WILSON_DIRAC

}

void cloverDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover cloverInv,
		      const cudaColorSpinorField *in, const int parity, const int dagger, 
		      const cudaColorSpinorField *x, const double &a,
		      const dim3 *blockDim, const dim3 *gridDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_CLOVER_DIRAC
  int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
    dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->Precision() != gauge.Precision())
    errorQuda("Mixing gauge and spinor precision not supported");

  if (in->Precision() != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  const void *xv = x ? x->V() : 0;
  const void *xn = x ? x->Norm() : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new CloverDslashCuda<double2, double2, double2>((double2*)out->V(), (float*)out->Norm(), (double2*)gauge0, 
							     (double2*)gauge1, gauge.Reconstruct(), (double2*)cloverP, 
							     (float*)cloverNormP, (double2*)in->V(), (float*)in->Norm(),
							     (double2*)xv, (float*)xn, a, dagger, in->Bytes(), in->NormBytes());
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    dslash = new CloverDslashCuda<float4, float4, float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, 
							  (float4*)gauge1, gauge.Reconstruct(), (float4*)cloverP, 
							  (float*)cloverNormP, (float4*)in->V(), (float*)in->Norm(), 
							  (float4*)xv, (float*)xn, a, dagger, in->Bytes(), in->NormBytes());
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    dslash = new CloverDslashCuda<short4, short4, short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, 
							  (short4*)gauge1, gauge.Reconstruct(), (short4*)cloverP, 
							  (float*)cloverNormP, (short4*)in->V(), (float*)in->Norm(), 
							  (short4*)xv, (float*)xn, a, dagger, in->Bytes(), in->NormBytes());
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), 
	     in->GhostFace(), blockDim, gridDim);

  delete dslash;
  unbindGaugeTex(gauge);
  unbindCloverTex(cloverInv);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}


void twistedMassDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			   const cudaColorSpinorField *in, const int parity, const int dagger, 
			   const cudaColorSpinorField *x, const double &kappa, const double &mu, 
			   const double &a, const dim3 *blockDim, const dim3 *gridDim, const int *commOverride) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_TWISTED_MASS_DIRAC
  int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
    dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->Precision() != gauge.Precision())
    errorQuda("Mixing gauge and spinor precision not supported");

  const void *xv = x ? x->V() : 0;
  const void *xn = x ? x->Norm() : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new TwistedDslashCuda<double2,double2>((double2*)out->V(), (float*)out->Norm(), (double2*)gauge0, 
						    (double2*)gauge1, gauge.Reconstruct(), (double2*)in->V(), 
						    (float*)in->Norm(), (double2*)xv, (float*)xn, 
						    kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    dslash = new TwistedDslashCuda<float4,float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, (float4*)gauge1, 
						  gauge.Reconstruct(), (float4*)in->V(), (float*)in->Norm(), 
						  (float4*)xv, (float*)xn, kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    dslash = new TwistedDslashCuda<short4,short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, (short4*)gauge1, 
						  gauge.Reconstruct(), (short4*)in->V(), (float*)in->Norm(), 
						  (short4*)xv, (float*)xn, kappa, mu, a, dagger, in->Bytes(), in->NormBytes());
    
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), 
	     in->GhostFace(), blockDim, gridDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif

}

void domainWallDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, 
			  const cudaColorSpinorField *in, const int parity, const int dagger, 
			  const cudaColorSpinorField *x, const double &m_f, const double &k2,
			  const dim3 *blockDim, const dim3 *gridDim) {

  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef MULTI_GPU
  errorQuda("Multi-GPU domain wall not implemented\n");
#endif

  dslashParam.parity = parity;
  dslashParam.threads = in->Volume();

#ifdef GPU_DOMAIN_WALL_DIRAC
  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (in->Precision() != gauge.Precision())
    errorQuda("Mixing gauge and spinor precision not supported");

  const void *xv = x ? x->V() : 0;
  const void *xn = x ? x->Norm() : 0;

  DslashCuda *dslash = 0;
  size_t regSize = sizeof(float);

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslash = new DomainWallDslashCuda<double2,double2>((double2*)out->V(), (float*)out->Norm(), (double2*)gauge0, (double2*)gauge1, 
						       gauge.Reconstruct(), (double2*)in->V(), (float*)in->Norm(), (double2*)xv, 
						       (float*)xn, m_f, k2, dagger, in->Bytes(), in->NormBytes());
    regSize = sizeof(double);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    dslash = new DomainWallDslashCuda<float4,float4>((float4*)out->V(), (float*)out->Norm(), (float4*)gauge0, (float4*)gauge1, 
						     gauge.Reconstruct(), (float4*)in->V(), (float*)in->Norm(), (float4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->Bytes(), in->NormBytes());
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    dslash = new DomainWallDslashCuda<short4,short4>((short4*)out->V(), (float*)out->Norm(), (short4*)gauge0, (short4*)gauge1, 
						     gauge.Reconstruct(), (short4*)in->V(), (float*)in->Norm(), (short4*)xv, 
						     (float*)xn, m_f, k2, dagger, in->Bytes(), in->NormBytes());
  }

  dslashCuda(*dslash, regSize, parity, dagger, in->Volume(), 
	     in->GhostFace(), blockDim, gridDim);

  delete dslash;
  unbindGaugeTex(gauge);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Domain wall dslash has not been built");
#endif

}


template <typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int* Vsh, const int* dims,
			 const int length, const int ghost_length, const dim3 *blockDim) {
    
  // calculate grid based on 4d volume
  dim3 interiorGridDim( (dslashParam.threads + blockDim[0].x -1)/blockDim[0].x, 1, 1);
  dim3 exteriorGridDim[4]  = {
    dim3((6*Vsh[0] + blockDim[1].x -1)/blockDim[1].x, 1, 1),
    dim3((6*Vsh[1] + blockDim[2].x -1)/blockDim[2].x, 1, 1),
    dim3((6*Vsh[2] + blockDim[3].x -1)/blockDim[3].x, 1, 1),
    dim3((6*Vsh[3] + blockDim[4].x -1)/blockDim[4].x, 1, 1)
  };

  // number of sources is set by the fifth dimension
  const int Nsrc = dims[4];  

#ifdef MULTI_GPU
  if (Nsrc > 1) errorQuda("Multi-source dslash does not yet support multi-gpu");
#endif

#ifdef PARALLEL_DIR    
  int blockz = 2;
#else
  int blockz = 1;
#endif

  dim3 blockDim2d(blockDim[0].x, Nsrc, blockz);

  size_t regSize = bindSpinorTex_mg(length, ghost_length, in, inNorm, x, xNorm);
  int shared_bytes = blockDim[0].x*6*regSize*Nsrc;
  
  dslashParam.kernel_type = INTERIOR_KERNEL;

#ifdef MULTI_GPU
  // wait for any previous outstanding dslashes to finish
  hipStreamWaitEvent(0, dslashEnd, 0);

  // Gather from source spinor
  for(int dir = 3; dir >=0 ; dir--){
    if (!dslashParam.commDim[dir]) continue;
    face->exchangeFacesStart(*inSpinor, 1-parity, dagger, dir, streams);
  }
#endif

  /*  printf("Launching (%d, %d, %d) threads on (%d, %d, %d) grid\n", 
	 blockDim2d.x, blockDim2d.y, blockDim2d.z, 
	 interiorGridDim.x, interiorGridDim.y, interiorGridDim.z);*/

  STAGGERED_DSLASH(interiorGridDim, blockDim2d, shared_bytes, streams[Nstream-1], dslashParam,
		   out, outNorm, fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a); 

  //if (!dslashTuning) checkCudaError();

#ifdef MULTI_GPU

  for(int i=3 ;i >= 0;i--){
    if (!dslashParam.commDim[i]) continue;

    // Finish gather and start comms
    face->exchangeFacesComms(i);
    // Wait for comms to finish, and scatter into the end zone
    face->exchangeFacesWait(*inSpinor, dagger,i);    

    // Record the end of the scattering
    hipEventRecord(scatterEvent[2*i], streams[2*i]);
    hipEventRecord(scatterEvent[2*i+1], streams[2*i+1]);
  }

  for(int i=3 ;i >= 0;i--){
    if(!dslashParam.commDim[i]) continue;

    shared_bytes = blockDim[i+1].x*6*regSize;

    //hipStreamSynchronize(streams[2*i]);
    //hipStreamSynchronize(streams[2*i + 1]);
    dslashParam.kernel_type = static_cast<KernelType>(i);
    dslashParam.tOffset =  dims[i]-6;
    dslashParam.threads = 6*Vsh[i];
    hipStreamWaitEvent(streams[Nstream-1], scatterEvent[2*i], 0);
    hipStreamWaitEvent(streams[Nstream-1], scatterEvent[2*i+1], 0);
    STAGGERED_DSLASH(exteriorGridDim[i], blockDim[i+1], shared_bytes, streams[Nstream-1], dslashParam,
		     out, outNorm, fatGauge0, fatGauge1, longGauge0, longGauge1, in, inNorm, x, xNorm, a);
    if (!dslashTuning) checkCudaError();
  }

  hipEventRecord(dslashEnd, streams[Nstream-1]);
  //hipStreamSynchronize(streams[Nstream-1]);

#endif
}

void staggeredDslashCuda(cudaColorSpinorField *out, const cudaGaugeField &fatGauge, 
			 const cudaGaugeField &longGauge, const cudaColorSpinorField *in,
			 const int parity, const int dagger, const cudaColorSpinorField *x,
			 const double &k, const dim3 *block, const dim3 *grid, const int *commOverride)
{
  
  inSpinor = (cudaColorSpinorField*)in; // EVIL

#ifdef GPU_STAGGERED_DIRAC

  dslashParam.parity = parity;
  dslashParam.threads = in->Volume() / in->X(4); // only the 4d volume is used here (grid dims + thread exit)
  
  int Npad = (in->Ncolor()*in->Nspin()*2)/in->FieldOrder(); // SPINOR_HOP in old code
  for(int i=0;i<4;i++){
    dslashParam.ghostDim[i] = commDimPartitioned(i); // determines whether to use regular or ghost indexing at boundary
    dslashParam.ghostOffset[i] = Npad*(in->GhostOffset(i) + in->Stride());
    dslashParam.ghostNormOffset[i] = in->GhostNormOffset(i) + in->Stride();
    dslashParam.commDim[i] = (!commOverride[i]) ? 0 : commDimPartitioned(i); // switch off comms if override = 0
  }
  void *fatGauge0, *fatGauge1;
  void* longGauge0, *longGauge1;
  bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
  bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
  
  if (in->Precision() != fatGauge.Precision() || in->Precision() != longGauge.Precision()){
    errorQuda("Mixing gauge and spinor precision not supported"
	      "(precision=%d, fatlinkGauge.precision=%d, longGauge.precision=%d",
	      in->Precision(), fatGauge.Precision(), longGauge.Precision());
  }
  
  const void *xv = x ? x->V() : 0;
  const void *xn = x ? x->Norm() : 0;
  
  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    staggeredDslashCuda((double2*)out->V(), (float*)out->Norm(), (double2*)fatGauge0, (double2*)fatGauge1,
			(double2*)longGauge0, (double2*)longGauge1, longGauge.Reconstruct(), 
			(double2*)in->V(), (float*)in->Norm(), parity, dagger, 
			(double2*)xv, (float*)x, k, in->Volume(), in->GhostFace(), 
			in->X(), in->Length(), in->GhostLength(), block);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    staggeredDslashCuda((float2*)out->V(), (float*)out->Norm(), (float2*)fatGauge0, (float2*)fatGauge1,
			  (float4*)longGauge0, (float4*)longGauge1, longGauge.Reconstruct(), 
			(float2*)in->V(), (float*)in->Norm(), parity, dagger, 
			(float2*)xv, (float*)xn, k, in->Volume(), in->GhostFace(), 
			in->X(), in->Length(), in->GhostLength(), block);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {	
    staggeredDslashCuda((short2*)out->V(), (float*)out->Norm(), (short2*)fatGauge0, (short2*)fatGauge1,
			(short4*)longGauge0, (short4*)longGauge1, longGauge.Reconstruct(), 
			(short2*)in->V(), (float*)in->Norm(), parity, dagger, 
			(short2*)xv, (float*)xn, k, in->Volume(), in->GhostFace(), 
			in->X(), in->Length(), in->GhostLength(), block);
  }
    
  if (!dslashTuning) checkCudaError();

#else
  errorQuda("Staggered dslash has not been built");
#endif  
}


template <typename spinorFloat, typename cloverFloat>
void cloverCuda(spinorFloat *out, float *outNorm, const cloverFloat *clover,
		const float *cloverNorm, const spinorFloat *in, const float *inNorm, 
		const size_t bytes, const size_t norm_bytes, const dim3 blockDim, const dim3 gridDim)
{
  //dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  int shared_bytes = blockDim.x*CLOVER_SHARED_FLOATS_PER_THREAD*bindSpinorTex(bytes, norm_bytes, in, inNorm);
  cloverKernel<<<gridDim, blockDim, shared_bytes>>> 
    (out, outNorm, clover, cloverNorm, in, inNorm, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void cloverCuda(cudaColorSpinorField *out, const cudaGaugeField &gauge, const FullClover clover, 
		const cudaColorSpinorField *in, const int parity, const dim3 &blockDim, const dim3 &gridDim) {

  dslashParam.parity = parity;
  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->Volume();

#ifdef GPU_CLOVER_DIRAC
  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (in->Precision() != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverCuda((double2*)out->V(), (float*)out->Norm(), (double2*)cloverP, 
	       (float*)cloverNormP, (double2*)in->V(), (float*)in->Norm(), 
	       in->Bytes(), in->NormBytes(), blockDim, gridDim);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    cloverCuda((float4*)out->V(), (float*)out->Norm(), (float4*)cloverP, 
	       (float*)cloverNormP, (float4*)in->V(), (float*)in->Norm(),
	       in->Bytes(), in->NormBytes(), blockDim, gridDim);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    cloverCuda((short4*)out->V(), (float*)out->Norm(), (short4*)cloverP, 
	       (float*)cloverNormP, (short4*)in->V(), (float*)in->Norm(), 
	       in->Bytes(), in->NormBytes(), blockDim, gridDim);
  }
  unbindCloverTex(clover);

  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Clover dslash has not been built");
#endif

}
// FIXME: twist kernel cannot be issued asynchronously because of texture unbinding
template <typename spinorFloat>
void twistGamma5Cuda(spinorFloat *out, float *outNorm, const spinorFloat *in, 
		     const float *inNorm, const int dagger, const double &kappa, 
		     const double &mu, const size_t bytes, const size_t norm_bytes, 
		     const QudaTwistGamma5Type twist, dim3 blockDim, dim3 gridDim)
{
  //  dim3 gridDim( (dslashParam.threads+blockDim.x-1) / blockDim.x, 1, 1);

  double a=0.0, b=0.0;
  setTwistParam(a, b, kappa, mu, dagger, twist);

  bindSpinorTex(bytes, norm_bytes, in, inNorm);
  twistGamma5Kernel<<<gridDim, blockDim, 0>>> (out, outNorm, a, b, dslashParam);
  unbindSpinorTex(in, inNorm);
}

void twistGamma5Cuda(cudaColorSpinorField *out, const cudaColorSpinorField *in,
		     const int dagger, const double &kappa, const double &mu,
		     const QudaTwistGamma5Type twist, const dim3 &block, const dim3 &grid) {

  dslashParam.tOffset = 0;
  dslashParam.tMul = 1;
  dslashParam.threads = in->Volume();

#ifdef GPU_TWISTED_MASS_DIRAC
  if (in->Precision() == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
<<<<<<< HEAD
    twistGamma5Cuda((double2*)out->V(), (float*)out->Norm(), 
		    (double2*)in->V(), (float*)in->Norm(), 
		    dagger, kappa, mu, in->Bytes(), 
		    in->NormBytes(), twist, block, grid);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (in->Precision() == QUDA_SINGLE_PRECISION) {
    twistGamma5Cuda((float4*)out->V(), (float*)out->Norm(),
		    (float4*)in->V(), (float*)in->Norm(), 
		    dagger, kappa, mu, in->Bytes(), 
		    in->NormBytes(), twist, block, grid);
  } else if (in->Precision() == QUDA_HALF_PRECISION) {
    twistGamma5Cuda((short4*)out->V(), (float*)out->Norm(),
		    (short4*)in->V(), (float*)in->Norm(), 
		    dagger, kappa, mu, in->Bytes(), 
		    in->NormBytes(), twist, block, grid);
  }
  if (!dslashTuning) checkCudaError();
#else
  errorQuda("Twisted mass dslash has not been built");
#endif // GPU_TWISTED_MASS_DIRAC
}


#include "misc_helpers.cu"


#if defined(GPU_FATLINK)||defined(GPU_GAUGE_FORCE)|| defined(GPU_FERMION_FORCE)
#include <force_common.h>
#include "force_kernel_common.cu"
#endif

#ifdef GPU_FATLINK
#include "llfat_quda.cu"
#endif

#ifdef GPU_GAUGE_FORCE
#include "gauge_force_quda.cu"
#endif

#ifdef GPU_FERMION_FORCE
#include "fermion_force_quda.cu"
#endif
