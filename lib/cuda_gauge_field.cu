#include "hip/hip_runtime.h"
#include <string.h>
#include <gauge_field.h>
#include <face_quda.h>
#include <typeinfo>
#include <misc_helpers.h>
#include <blas_quda.h>

namespace quda {

  cudaGaugeField::cudaGaugeField(const GaugeFieldParam &param) :
    GaugeField(param), gauge(0), even(0), odd(0), backed_up(false)
  {
    if (order == QUDA_QDP_GAUGE_ORDER) errorQuda("QDP ordering not supported");
    
    if(create != QUDA_NULL_FIELD_CREATE &&  
       create != QUDA_ZERO_FIELD_CREATE && 
       create != QUDA_REFERENCE_FIELD_CREATE){
      errorQuda("ERROR: create type(%d) not supported yet\n", create);
    }
  
    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      gauge = device_malloc(bytes);  
      if (create == QUDA_ZERO_FIELD_CREATE) hipMemset(gauge, 0, bytes);
    } else { // for reference fields (e.g., external fields) we need to do the ghost exchange
      gauge = param.gauge;
      exchangeGhost();
    }

    even = gauge;
    odd = (char*)gauge + bytes/2; 

#ifdef USE_TEXTURE_OBJECTS
    createTexObject(evenTex, even);
    createTexObject(oddTex, odd);
#endif
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaGaugeField::createTexObject(hipTextureObject_t &tex, void *field) {
    // create the texture for the field components
    hipChannelFormatDesc desc;
    memset(&desc, 0, sizeof(hipChannelFormatDesc));
    if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
    else desc.f = hipChannelFormatKindSigned; // half is short, double is int2

    // always four components regardless of precision
    if (precision == QUDA_DOUBLE_PRECISION) {
      desc.x = 8*sizeof(int);
      desc.y = 8*sizeof(int);
      desc.z = 8*sizeof(int);
      desc.w = 8*sizeof(int);
    } else {
      desc.x = 8*precision;
      desc.y = 8*precision;
      desc.z = (reconstruct == 18) ? 0 : 8*precision; // float2 or short2 for 18 reconstruct
      desc.w = (reconstruct == 18) ? 0 : 8*precision;
    }

    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = field;
    resDesc.res.linear.desc = desc;
    resDesc.res.linear.sizeInBytes = bytes/2;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
    else texDesc.readMode = hipReadModeElementType;

    hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
    checkCudaError();
  }

  void cudaGaugeField::destroyTexObject() {
    hipDestroyTextureObject(evenTex);
    hipDestroyTextureObject(oddTex);
    checkCudaError();
  }
#endif

  cudaGaugeField::~cudaGaugeField()
  {
#ifdef USE_TEXTURE_OBJECTS
    destroyTexObject();
#endif

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (gauge) device_free(gauge);
    }
  }

  // This does the exchange of the gauge field ghost zone and places it
  // into the ghost array.
  void cudaGaugeField::exchangeGhost() {
    if (ghostExchange) return;

    void *ghost[QUDA_MAX_DIM];
    void *send[QUDA_MAX_DIM];
    for (int d=0; d<nDim; d++) {
      ghost[d] = device_malloc(nFace*surface[d]*reconstruct*precision);
      send[d] = device_malloc(nFace*surface[d]*reconstruct*precision);
    }

    // get the links into contiguous buffers
    extractGaugeGhost(*this, send);

    // communicate between nodes
    FaceBuffer faceBuf(x, nDim, reconstruct, nFace, precision);
    faceBuf.exchangeLink(ghost, send, QUDA_CUDA_FIELD_LOCATION);

    // copy from ghost into the padded region in gauge
    copyGenericGauge(*this, *this, QUDA_CUDA_FIELD_LOCATION, 0, 0, 0, ghost);

    for (int d=0; d<nDim; d++) {
      device_free(send[d]);
      device_free(ghost[d]);
    }

    ghostExchange = true;
  }

  void cudaGaugeField::setGauge(void *gauge_)
  {
    if(create != QUDA_REFERENCE_FIELD_CREATE) {
      errorQuda("Setting gauge pointer is only allowed when create="
		"QUDA_REFERENCE_FIELD_CREATE type\n");
    }
    gauge = gauge_;
  }

  void cudaGaugeField::copy(const GaugeField &src) {

    if (geometry != QUDA_VECTOR_GEOMETRY) errorQuda("Only vector geometry is supported");
    checkField(src);

    if (link_type == QUDA_ASQTAD_FAT_LINKS) {
      fat_link_max = src.LinkMax();
      if (precision == QUDA_HALF_PRECISION && fat_link_max == 0.0) 
	errorQuda("fat_link_max has not been computed");
    }
    
    if (typeid(src) == typeid(cudaGaugeField)) {
      // copy field and ghost zone into this field
      copyGenericGauge(*this, src, QUDA_CUDA_FIELD_LOCATION, gauge, 
		       static_cast<const cudaGaugeField&>(src).gauge);

    } else if (typeid(src) == typeid(cpuGaugeField)) {
      LatticeField::resizeBuffer(bytes);

      // copy field and ghost zone into bufferPinned
      copyGenericGauge(*this, src, QUDA_CPU_FIELD_LOCATION, bufferPinned, 
		       static_cast<const cpuGaugeField&>(src).gauge); 

      // this copies over both even and odd
      hipMemcpy(gauge, bufferPinned, bytes, hipMemcpyHostToDevice);
    } else {
      errorQuda("Invalid gauge field type");
    }

    checkCudaError();
  }

  void cudaGaugeField::loadCPUField(const cpuGaugeField &cpu, const QudaFieldLocation &pack_location)
  {
    if (geometry != QUDA_VECTOR_GEOMETRY) errorQuda("Only vector geometry is supported");

    if (pack_location == QUDA_CUDA_FIELD_LOCATION) {
      errorQuda("Not implemented");
    } else if (pack_location == QUDA_CPU_FIELD_LOCATION) {
      copy(cpu);
    } else {
      errorQuda("Invalid pack location %d", pack_location);
    }

  }
  
  /*
    Copies the device gauge field to the host.
    - no reconstruction support
    - device data is always Float2 ordered
    - host data is a 1-dimensional array (MILC ordered)
    - no support for half precision
    - input and output precisions must match
  */
  template<typename FloatN, typename Float>
  static void storeGaugeField(Float* cpuGauge, FloatN *gauge, int bytes, int volumeCB, 
			      int stride, QudaPrecision prec) 
  {  
    hipStream_t streams[2];
    for (int i=0; i<2; i++) hipStreamCreate(&streams[i]);
  
    FloatN *even = gauge;
    FloatN *odd = (FloatN*)((char*)gauge + bytes/2);

    size_t datalen = 4*2*volumeCB*gaugeSiteSize*sizeof(Float); // both parities
    void *unpacked = device_malloc(datalen);
    void *unpackedEven = unpacked;
    void *unpackedOdd = (char*)unpacked + datalen/2;
  
    //unpack even data kernel
    link_format_gpu_to_cpu((void*)unpackedEven, (void*)even, volumeCB, stride, prec, streams[0]);
#ifdef GPU_DIRECT
    hipMemcpyAsync(cpuGauge, unpackedEven, datalen/2, hipMemcpyDeviceToHost, streams[0]);
#else
    hipMemcpy(cpuGauge, unpackedEven, datalen/2, hipMemcpyDeviceToHost);
#endif
  
    //unpack odd data kernel
    link_format_gpu_to_cpu((void*)unpackedOdd, (void*)odd, volumeCB, stride, prec, streams[1]);
#ifdef GPU_DIRECT
    hipMemcpyAsync(cpuGauge + 4*volumeCB*gaugeSiteSize, unpackedOdd, datalen/2, hipMemcpyDeviceToHost, streams[1]);  
    for(int i=0; i<2; i++) hipStreamSynchronize(streams[i]);
#else
    hipMemcpy(cpuGauge + 4*volumeCB*gaugeSiteSize, unpackedOdd, datalen/2, hipMemcpyDeviceToHost);  
#endif
  
    device_free(unpacked);
    for(int i=0; i<2; i++) hipStreamDestroy(streams[i]);
  }

  void cudaGaugeField::saveCPUField(cpuGaugeField &cpu, const QudaFieldLocation &pack_location) const
  {
    if (geometry != QUDA_VECTOR_GEOMETRY) errorQuda("Only vector geometry is supported");

    // FIXME use the generic copying for the below copying
    // do device-side reordering then copy
    if (pack_location == QUDA_CUDA_FIELD_LOCATION) {
      // check parameters are suitable for device-side packing
      if (precision != cpu.Precision())
	errorQuda("cpu precision %d and cuda precision %d must be the same", 
		  cpu.Precision(), precision);

      if (reconstruct != QUDA_RECONSTRUCT_NO) errorQuda("Only no reconstruction supported");
      if (order != QUDA_FLOAT2_GAUGE_ORDER) errorQuda("Only QUDA_FLOAT2_GAUGE_ORDER supported");
      if (cpu.Order() != QUDA_MILC_GAUGE_ORDER) errorQuda("Only QUDA_MILC_GAUGE_ORDER supported");

      if (precision == QUDA_DOUBLE_PRECISION){
	storeGaugeField((double*)cpu.gauge, (double2*)gauge, bytes, volumeCB, stride, precision);
      } else if (precision == QUDA_SINGLE_PRECISION){
	storeGaugeField((float*)cpu.gauge, (float2*)gauge, bytes, volumeCB, stride, precision);
      } else {
	errorQuda("Half precision not supported");
      }

    } else if (pack_location == QUDA_CPU_FIELD_LOCATION) { // do copy then host-side reorder
    
      resizeBuffer(bytes);

      // this copies over both even and odd
      hipMemcpy(bufferPinned, gauge, bytes, hipMemcpyDeviceToHost);
      checkCudaError();

      copyGenericGauge(cpu, *this, QUDA_CPU_FIELD_LOCATION, cpu.gauge, bufferPinned);
    } else {
      errorQuda("Invalid pack location %d", pack_location);
    }

  }

  void cudaGaugeField::backup() const {
    if (backed_up) errorQuda("Gauge field already backed up");
    backup_h = new char[bytes];
    hipMemcpy(backup_h, gauge, bytes, hipMemcpyDeviceToHost);
    checkCudaError();
    backed_up = true;
  }

  void cudaGaugeField::restore() {
    if (!backed_up) errorQuda("Cannot restore since not backed up");
    hipMemcpy(gauge, backup_h, bytes, hipMemcpyHostToDevice);
    delete []backup_h;
    checkCudaError();
    backed_up = false;
  }

  // Return the L2 norm squared of the gauge field
  double norm2(const cudaGaugeField &a) {
  
    int spin = 0;
    switch (a.Geometry()) {
    case QUDA_SCALAR_GEOMETRY:
      spin = 1;
      break;
    case QUDA_VECTOR_GEOMETRY:
      spin = a.Ndim();
      break;
    case QUDA_TENSOR_GEOMETRY:
      spin = a.Ndim() * (a.Ndim()-1);
      break;
    default:
      errorQuda("Unsupported field geometry %d", a.Geometry());
    }

    if (a.Precision() == QUDA_HALF_PRECISION) 
      errorQuda("Casting a cudaGaugeField into cudaColorSpinorField not possible in half precision");
    
    if (a.Reconstruct() == QUDA_RECONSTRUCT_13 || a.Reconstruct() == QUDA_RECONSTRUCT_9)
      errorQuda("Unsupported field reconstruct %d", a.Reconstruct());
      


    ColorSpinorParam spinor_param;
    spinor_param.nColor = a.Reconstruct()/2;
    spinor_param.nSpin = a.Ndim();
    spinor_param.nDim = spin;
    for (int d=0; d<a.Ndim(); d++) spinor_param.x[d] = a.X()[d];
    spinor_param.precision = a.Precision();
    spinor_param.pad = a.Pad();
    spinor_param.siteSubset = QUDA_FULL_SITE_SUBSET;
    spinor_param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
    spinor_param.fieldOrder = (QudaFieldOrder)a.FieldOrder();
    spinor_param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
    spinor_param.create = QUDA_REFERENCE_FIELD_CREATE;
    spinor_param.v = (void*)a.Gauge_p();
    cudaColorSpinorField b(spinor_param);
    return norm2(b);
  }

} // namespace quda
