#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <typeinfo>

#include <color_spinor_field.h>
#include <blas_quda.h>

#include <string.h>
#include <iostream>
#include <misc_helpers.h>
#include <face_quda.h>
#include <dslash_quda.h>

#ifdef DEVICE_PACK
#define REORDER_LOCATION QUDA_CUDA_FIELD_LOCATION
#else
#define REORDER_LOCATION QUDA_CPU_FIELD_LOCATION
#endif

int zeroCopy = 0;

namespace quda {

  int cudaColorSpinorField::initGhostFaceBuffer = 0;
  void* cudaColorSpinorField::ghostFaceBuffer; //gpu memory
  void* cudaColorSpinorField::fwdGhostFaceBuffer[QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  void* cudaColorSpinorField::backGhostFaceBuffer[QUDA_MAX_DIM]; //pointers to ghostFaceBuffer
  QudaPrecision cudaColorSpinorField::facePrecision; 

  /*cudaColorSpinorField::cudaColorSpinorField() : 
    ColorSpinorField(), v(0), norm(0), alloc(false), init(false) {

    }*/

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorParam &param) : 
    ColorSpinorField(param), alloc(false), init(true), texInit(false) {

    // this must come before create
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      v = param.v;
      norm = param.norm;
    }

    create(param.create);

    if  (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // dp nothing
    } else if (param.create == QUDA_COPY_FIELD_CREATE){
      errorQuda("not implemented");
    }
    checkCudaError();
  }

  cudaColorSpinorField::cudaColorSpinorField(const cudaColorSpinorField &src) : 
    ColorSpinorField(src), alloc(false), init(true), texInit(false) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
  }

  // creates a copy of src, any differences defined in param
  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src, 
					     const ColorSpinorParam &param) :
    ColorSpinorField(src), alloc(false), init(true), texInit(false) {  

    // can only overide if we are not using a reference or parity special case
    if (param.create != QUDA_REFERENCE_FIELD_CREATE || 
	(param.create == QUDA_REFERENCE_FIELD_CREATE && 
	 src.SiteSubset() == QUDA_FULL_SITE_SUBSET && 
	 param.siteSubset == QUDA_PARITY_SITE_SUBSET && 
	 typeid(src) == typeid(cudaColorSpinorField) ) ) {
      reset(param);
    } else {
      errorQuda("Undefined behaviour"); // else silent bug possible?
    }

    // This must be set before create is called
    if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      if (typeid(src) == typeid(cudaColorSpinorField)) {
	v = (void*)src.V();
	norm = (void*)src.Norm();
      } else {
	errorQuda("Cannot reference a non-cuda field");
      }
    }

    create(param.create);

    if (param.create == QUDA_NULL_FIELD_CREATE) {
      // do nothing
    } else if (param.create == QUDA_ZERO_FIELD_CREATE) {
      zero();
    } else if (param.create == QUDA_COPY_FIELD_CREATE) {
      copySpinorField(src);
    } else if (param.create == QUDA_REFERENCE_FIELD_CREATE) {
      // do nothing
    } else {
      errorQuda("CreateType %d not implemented", param.create);
    }

    clearGhostPointers();
  }

  cudaColorSpinorField::cudaColorSpinorField(const ColorSpinorField &src) 
    : ColorSpinorField(src), alloc(false), init(true), texInit(false) {
    create(QUDA_COPY_FIELD_CREATE);
    copySpinorField(src);
    clearGhostPointers();
  }

  ColorSpinorField& cudaColorSpinorField::operator=(const ColorSpinorField &src) {
    if (typeid(src) == typeid(cudaColorSpinorField)) {
      *this = (dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cpuColorSpinorField)) {
      *this = (dynamic_cast<const cpuColorSpinorField&>(src));
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cudaColorSpinorField &src) {
    if (&src != this) {
      // keep current attributes unless unset
      if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
	destroy();
	ColorSpinorField::operator=(src);
	create(QUDA_COPY_FIELD_CREATE);
      }
      copySpinorField(src);
    }
    return *this;
  }

  cudaColorSpinorField& cudaColorSpinorField::operator=(const cpuColorSpinorField &src) {
    // keep current attributes unless unset
    if (!ColorSpinorField::init) { // note this will turn a reference field into a regular field
      destroy();
      ColorSpinorField::operator=(src);
      create(QUDA_COPY_FIELD_CREATE);
    }
    loadSpinorField(src);
    return *this;
  }

  cudaColorSpinorField::~cudaColorSpinorField() {
    destroy();
  }

  bool cudaColorSpinorField::isNative() const {

    if (precision == QUDA_DOUBLE_PRECISION) {
      if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
    } else if (precision == QUDA_SINGLE_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    } else if (precision == QUDA_HALF_PRECISION) {
      if (nSpin == 4) {
	if (fieldOrder == QUDA_FLOAT4_FIELD_ORDER) return true;
      } else if (nSpin == 1) {
	if (fieldOrder == QUDA_FLOAT2_FIELD_ORDER) return true;
      }
    }

    return false;
  }

  void cudaColorSpinorField::create(const QudaFieldCreate create) {

    if (siteSubset == QUDA_FULL_SITE_SUBSET && siteOrder != QUDA_EVEN_ODD_SITE_ORDER) {
      errorQuda("Subset not implemented");
    }

    //FIXME: This addition is temporary to ensure we have the correct
    //field order for a given precision
    //if (precision == QUDA_DOUBLE_PRECISION) fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
    //else fieldOrder = (nSpin == 4) ? QUDA_FLOAT4_FIELD_ORDER : QUDA_FLOAT2_FIELD_ORDER;

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      v = device_malloc(bytes);
      if (precision == QUDA_HALF_PRECISION) {
	norm = device_malloc(norm_bytes);
      }
      alloc = true;
    }

    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      // create the associated even and odd subsets
      ColorSpinorParam param;
      param.siteSubset = QUDA_PARITY_SITE_SUBSET;
      param.nDim = nDim;
      memcpy(param.x, x, nDim*sizeof(int));
      param.x[0] /= 2; // set single parity dimensions
      param.create = QUDA_REFERENCE_FIELD_CREATE;
      param.v = v;
      param.norm = norm;
      even = new cudaColorSpinorField(*this, param);
      odd = new cudaColorSpinorField(*this, param);

      // need this hackery for the moment (need to locate the odd pointer half way into the full field)
      (dynamic_cast<cudaColorSpinorField*>(odd))->v = (void*)((char*)v + bytes/2);
      if (precision == QUDA_HALF_PRECISION) 
	(dynamic_cast<cudaColorSpinorField*>(odd))->norm = (void*)((char*)norm + norm_bytes/2);

#ifdef USE_TEXTURE_OBJECTS
      dynamic_cast<cudaColorSpinorField*>(even)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(even)->createTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->destroyTexObject();
      dynamic_cast<cudaColorSpinorField*>(odd)->createTexObject();
#endif
    }

    if (create != QUDA_REFERENCE_FIELD_CREATE) {
      if (siteSubset != QUDA_FULL_SITE_SUBSET) {
	zeroPad();
      } else {
	(dynamic_cast<cudaColorSpinorField*>(even))->zeroPad();
	(dynamic_cast<cudaColorSpinorField*>(odd))->zeroPad();
      }
    }

#ifdef USE_TEXTURE_OBJECTS
    createTexObject();
#endif

    checkCudaError();
  }

#ifdef USE_TEXTURE_OBJECTS
  void cudaColorSpinorField::createTexObject() {

    if (isNative()) {
      if (texInit) errorQuda("Already bound textures");
      
      // create the texture for the field components
      
      hipChannelFormatDesc desc;
      memset(&desc, 0, sizeof(hipChannelFormatDesc));
      if (precision == QUDA_SINGLE_PRECISION) desc.f = hipChannelFormatKindFloat;
      else desc.f = hipChannelFormatKindSigned; // half is short, double is int2
      
      // staggered fields in half and single are always two component
      if (nSpin == 1 && (precision == QUDA_HALF_PRECISION || precision == QUDA_SINGLE_PRECISION)) {
	desc.x = 8*precision;
	desc.y = 8*precision;
	desc.z = 0;
	desc.w = 0;
      } else { // all others are four component
	desc.x = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.y = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.z = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
	desc.w = (precision == QUDA_DOUBLE_PRECISION) ? 32 : 8*precision;
      }
      
      hipResourceDesc resDesc;
      memset(&resDesc, 0, sizeof(resDesc));
      resDesc.resType = hipResourceTypeLinear;
      resDesc.res.linear.devPtr = v;
      resDesc.res.linear.desc = desc;
      resDesc.res.linear.sizeInBytes = bytes;
      
      hipTextureDesc texDesc;
      memset(&texDesc, 0, sizeof(texDesc));
      if (precision == QUDA_HALF_PRECISION) texDesc.readMode = hipReadModeNormalizedFloat;
      else texDesc.readMode = hipReadModeElementType;
      
      hipCreateTextureObject(&tex, &resDesc, &texDesc, NULL);
      checkCudaError();
      
      // create the texture for the norm components
      if (precision == QUDA_HALF_PRECISION) {
	hipChannelFormatDesc desc;
	memset(&desc, 0, sizeof(hipChannelFormatDesc));
	desc.f = hipChannelFormatKindFloat;
	desc.x = 8*QUDA_SINGLE_PRECISION; desc.y = 0; desc.z = 0; desc.w = 0;
	
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypeLinear;
	resDesc.res.linear.devPtr = norm;
	resDesc.res.linear.desc = desc;
	resDesc.res.linear.sizeInBytes = norm_bytes;
	
	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.readMode = hipReadModeElementType;
	
	hipCreateTextureObject(&texNorm, &resDesc, &texDesc, NULL);
	checkCudaError();
      }
      
      texInit = true;
    }
  }

  void cudaColorSpinorField::destroyTexObject() {
    if (isNative() && texInit) {
      hipDestroyTextureObject(tex);
      if (precision == QUDA_HALF_PRECISION) hipDestroyTextureObject(texNorm);
      texInit = false;
      checkCudaError();
    }
  }
#endif

  void cudaColorSpinorField::destroy() {
    if (alloc) {
      device_free(v);
      if (precision == QUDA_HALF_PRECISION) device_free(norm);
      if (siteSubset == QUDA_FULL_SITE_SUBSET) {
	delete even;
	delete odd;
      }
      alloc = false;
    }

#ifdef USE_TEXTURE_OBJECTS
    destroyTexObject();
#endif

  }

  cudaColorSpinorField& cudaColorSpinorField::Even() const { 
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(even)); 
    }

    errorQuda("Cannot return even subset of %d subset", siteSubset);
    exit(-1);
  }

  cudaColorSpinorField& cudaColorSpinorField::Odd() const {
    if (siteSubset == QUDA_FULL_SITE_SUBSET) {
      return *(dynamic_cast<cudaColorSpinorField*>(odd)); 
    }

    errorQuda("Cannot return odd subset of %d subset", siteSubset);
    exit(-1);
  }

  // cuda's floating point format, IEEE-754, represents the floating point
  // zero as 4 zero bytes
  void cudaColorSpinorField::zero() {
    hipMemsetAsync(v, 0, bytes, streams[Nstream-1]);
    if (precision == QUDA_HALF_PRECISION) hipMemsetAsync(norm, 0, norm_bytes, streams[Nstream-1]);
  }


  void cudaColorSpinorField::zeroPad() {
    size_t pad_bytes = (stride - volume) * precision * fieldOrder;
    int Npad = nColor * nSpin * 2 / fieldOrder;
    for (int i=0; i<Npad; i++) {
      if (pad_bytes) hipMemset((char*)v + (volume + i*stride)*fieldOrder*precision, 0, pad_bytes);
    }
  }

  void cudaColorSpinorField::copy(const cudaColorSpinorField &src) {
    checkField(*this, src);
    copyCuda(*this, src);
  }

  void cudaColorSpinorField::copySpinorField(const ColorSpinorField &src) {
    
    // src is on the device and is native
    if (typeid(src) == typeid(cudaColorSpinorField) && 
	isNative() && dynamic_cast<const cudaColorSpinorField &>(src).isNative()) {
      copy(dynamic_cast<const cudaColorSpinorField&>(src));
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else if (typeid(src) == typeid(cpuColorSpinorField)) { // src is on the host
      loadSpinorField(src);
    } else {
      errorQuda("Unknown input ColorSpinorField %s", typeid(src).name());
    }
  } 

  void cudaColorSpinorField::loadSpinorField(const ColorSpinorField &src) {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(src) == typeid(cpuColorSpinorField)) {
      resizeBufferPinned(bytes + norm_bytes);
      memset(bufferPinned, 0, bytes+norm_bytes); // FIXME (temporary?) bug fix for padding

      copyGenericColorSpinor(*this, src, QUDA_CPU_FIELD_LOCATION, 
			     bufferPinned, 0, (char*)bufferPinned+bytes, 0);

      hipMemcpy(v, bufferPinned, bytes, hipMemcpyHostToDevice);
      hipMemcpy(norm, (char*)bufferPinned+bytes, norm_bytes, hipMemcpyHostToDevice);
    } else if (typeid(src) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *Src, *srcNorm;
      if (!zeroCopy) {
	resizeBufferDevice(src.Bytes()+src.NormBytes());
	Src = bufferDevice;
	srcNorm = (char*)bufferDevice + src.Bytes();	
	hipMemcpy(Src, src.V(), src.Bytes(), hipMemcpyHostToDevice);
	hipMemcpy(srcNorm, src.Norm(), src.NormBytes(), hipMemcpyHostToDevice);
      } else {
	resizeBufferPinned(src.Bytes()+src.NormBytes());
	memcpy(bufferPinned, src.V(), src.Bytes());
	memcpy((char*)bufferPinned+src.Bytes(), src.Norm(), src.NormBytes());

	hipHostGetDevicePointer(&Src, bufferPinned, 0);
	srcNorm = (void*)((char*)Src + src.Bytes());
      }

      hipMemset(v, 0, bytes); // FIXME (temporary?) bug fix for padding
      copyGenericColorSpinor(*this, src, QUDA_CUDA_FIELD_LOCATION, 0, Src, 0, srcNorm);
    }

    checkCudaError();
    return;
  }


  void cudaColorSpinorField::saveSpinorField(ColorSpinorField &dest) const {

    if (REORDER_LOCATION == QUDA_CPU_FIELD_LOCATION && 
	typeid(dest) == typeid(cpuColorSpinorField)) {
      resizeBufferPinned(bytes+norm_bytes);
      hipMemcpy(bufferPinned, v, bytes, hipMemcpyDeviceToHost);
      hipMemcpy((char*)bufferPinned+bytes, norm, norm_bytes, hipMemcpyDeviceToHost);

      copyGenericColorSpinor(dest, *this, QUDA_CPU_FIELD_LOCATION, 
			     0, bufferPinned, 0, (char*)bufferPinned+bytes);
    } else if (typeid(dest) == typeid(cudaColorSpinorField)) {
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION);
    } else {
      void *dst, *dstNorm;
      if (!zeroCopy) {
	resizeBufferDevice(dest.Bytes()+dest.NormBytes());
	dst = bufferDevice;
	dstNorm = (char*)bufferDevice+dest.Bytes();
      } else {
	resizeBufferPinned(dest.Bytes()+dest.NormBytes());
	hipHostGetDevicePointer(&dst, bufferPinned, 0);
	dstNorm = (char*)dst+dest.Bytes();
      }
      copyGenericColorSpinor(dest, *this, QUDA_CUDA_FIELD_LOCATION, dst, v, dstNorm, 0);

      if (!zeroCopy) {
	hipMemcpy(dest.V(), dst, dest.Bytes(), hipMemcpyDeviceToHost);
	hipMemcpy(dest.Norm(), dstNorm, dest.NormBytes(), hipMemcpyDeviceToHost);
      } else {
	memcpy(dest.V(), bufferPinned, dest.Bytes());
	memcpy(dest.Norm(), (char*)bufferPinned+dest.Bytes(), dest.NormBytes());
      }
    }

    checkCudaError();
    return;
  }

  void cudaColorSpinorField::allocateGhostBuffer(void) {
    int nFace = (nSpin == 1) ? 3 : 1; //3 faces for asqtad
    int Nint = nColor * nSpin * 2; // number of internal degrees of freedom
    if (nSpin == 4) Nint /= 2; // spin projection for Wilson

    // only allocate if not already allocated or precision is greater then previously
    if(initGhostFaceBuffer == 0 || precision > facePrecision){    

      if (initGhostFaceBuffer) device_free(ghostFaceBuffer); 

      // allocate a single contiguous buffer for the buffers
      size_t faceBytes = 0;
      for (int i=0; i<4; i++) {
	if(!commDimPartitioned(i)) continue;
	faceBytes += 2*nFace*ghostFace[i]*Nint*precision;
	// add extra space for the norms for half precision
	if (precision == QUDA_HALF_PRECISION) faceBytes += 2*nFace*ghostFace[i]*sizeof(float);
      }

      if (faceBytes > 0) {
	ghostFaceBuffer = device_malloc(faceBytes);
	initGhostFaceBuffer = 1;
	facePrecision = precision;
      }

    }

    size_t offset = 0;
    for (int i=0; i<4; i++) {
      if(!commDimPartitioned(i)) continue;
      
      backGhostFaceBuffer[i] = (void*)(((char*)ghostFaceBuffer) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
      
      fwdGhostFaceBuffer[i] = (void*)(((char*)ghostFaceBuffer) + offset);
      offset += nFace*ghostFace[i]*Nint*precision;
      if (precision == QUDA_HALF_PRECISION) offset += nFace*ghostFace[i]*sizeof(float);
    }   
    
  }


  void cudaColorSpinorField::freeGhostBuffer(void)
  {
    if (!initGhostFaceBuffer) return;
  
    device_free(ghostFaceBuffer); 

    for(int i=0;i < 4; i++){
      if(!commDimPartitioned(i)) continue;
      backGhostFaceBuffer[i] = NULL;
      fwdGhostFaceBuffer[i] = NULL;
    }
    initGhostFaceBuffer = 0;  
  }

  // pack the ghost zone into a contiguous buffer for communications
  void cudaColorSpinorField::packGhost(const QudaParity parity, const int dim, const QudaDirection dir, const int dagger, 
				       hipStream_t *stream, void *buffer) 
  {
    int face_num;
    if(dir == QUDA_BACKWARDS){
      face_num = 0;
    }else if(dir == QUDA_FORWARDS){
      face_num = 1;
    }else{
      face_num = 2;
    }
#ifdef MULTI_GPU
    void *packBuffer = buffer ? buffer : ghostFaceBuffer;
    packFace(packBuffer, *this, dagger, parity, dim, face_num, *stream); 
#else
    errorQuda("packGhost not built on single-GPU build");
#endif

  }

  void cudaColorSpinorField::packTwistedGhost(const QudaParity parity, const int dagger, 
					      double a, double b, hipStream_t *stream, void *buffer) 
  {
#ifdef MULTI_GPU
    void *packBuffer = buffer ? buffer : ghostFaceBuffer;
    packTwistedFace(packBuffer, *this, dagger, parity, a, b, *stream); 
#else
    errorQuda("packTwistedGhost not built on single-GPU build");
#endif

  }
 
  // send the ghost zone to the host
  void cudaColorSpinorField::sendGhost(void *ghost_spinor, const int dim, const QudaDirection dir,
				       const int dagger, hipStream_t *stream) {

#ifdef MULTI_GPU
    int Nvec = (nSpin == 1 || precision == QUDA_DOUBLE_PRECISION) ? 2 : 4;
    int nFace = (nSpin == 1) ? 3 : 1; //3 faces for asqtad
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    if (dim !=3 || getKernelPackT() || getTwistPack()) { // use kernels to pack into contiguous buffers then a single hipMemcpy

      size_t bytes = nFace*Nint*ghostFace[dim]*precision;
      if (precision == QUDA_HALF_PRECISION) bytes += nFace*ghostFace[dim]*sizeof(float);
      void* gpu_buf = 
	(dir == QUDA_BACKWARDS) ? this->backGhostFaceBuffer[dim] : this->fwdGhostFaceBuffer[dim];

      hipMemcpyAsync(ghost_spinor, gpu_buf, bytes, hipMemcpyDeviceToHost, *stream); 
    } else if(this->TwistFlavor() != QUDA_TWIST_NONDEG_DOUBLET){ // do multiple cudaMemcpys

      int Npad = Nint / Nvec; // number Nvec buffers we have
      int Nt_minus1_offset = (volume - nFace*ghostFace[3]); // N_t -1 = Vh-Vsh
      int offset = 0;
      if (nSpin == 1) {
	offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset;
      } else if (nSpin == 4) {    
	// !dagger: send lower components backwards, send upper components forwards
	// dagger: send upper components backwards, send lower components forwards
	bool upper = dagger ? true : false; // Fwd is !Back  
	if (dir == QUDA_FORWARDS) upper = !upper;
	int lower_spin_offset = Npad*stride;
	if (upper) offset = (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
	else offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : Nt_minus1_offset);
      }
    
      // QUDA Memcpy NPad's worth. 
      //  -- Dest will point to the right beginning PAD. 
      //  -- Each Pad has size Nvec*Vsh Floats. 
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + offset*Nvec*precision;
      size_t len = nFace*ghostFace[3]*Nvec*precision;     
      size_t spitch = stride*Nvec*precision;
      hipMemcpy2DAsync(dst, len, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
	int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + nFace*Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
	hipMemcpyAsync(dst, src, nFace*ghostFace[3]*sizeof(float), hipMemcpyDeviceToHost, *stream); 
      }
    }else{
      int flavorVolume = volume / 2;
      int flavorTFace  = ghostFace[3] / 2;
      int Npad = Nint / Nvec; // number Nvec buffers we have
      int flavor1_Nt_minus1_offset = (flavorVolume - flavorTFace);
      int flavor2_Nt_minus1_offset = (volume - flavorTFace);
      int flavor1_offset = 0;
      int flavor2_offset = 0;
      // !dagger: send lower components backwards, send upper components forwards
      // dagger: send upper components backwards, send lower components forwards
      bool upper = dagger ? true : false; // Fwd is !Back
      if (dir == QUDA_FORWARDS) upper = !upper;
      int lower_spin_offset = Npad*stride;//ndeg tm: stride=2*flavor_volume+pad
      if (upper){
        flavor1_offset = (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }else{
        flavor1_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? 0 : flavor1_Nt_minus1_offset);
        flavor2_offset = lower_spin_offset + (dir == QUDA_BACKWARDS ? flavorVolume : flavor2_Nt_minus1_offset);
      }

      // QUDA Memcpy NPad's worth.
      //  -- Dest will point to the right beginning PAD.
      //  -- Each Pad has size Nvec*Vsh Floats.
      //  --  There is Nvec*Stride Floats from the start of one PAD to the start of the next

      void *dst = (char*)ghost_spinor;
      void *src = (char*)v + flavor1_offset*Nvec*precision;
      size_t len = flavorTFace*Nvec*precision;
      size_t spitch = stride*Nvec*precision;//ndeg tm: stride=2*flavor_volume+pad
      size_t dpitch = 2*len;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);
      dst = (char*)ghost_spinor+len;
      src = (char*)v + flavor2_offset*Nvec*precision;
      hipMemcpy2DAsync(dst, dpitch, src, spitch, len, Npad, hipMemcpyDeviceToHost, *stream);

      if (precision == QUDA_HALF_PRECISION) {
        int Nt_minus1_offset = (flavorVolume - flavorTFace);
        int norm_offset = (dir == QUDA_BACKWARDS) ? 0 : Nt_minus1_offset*sizeof(float);
	void *dst = (char*)ghost_spinor + Nint*ghostFace[3]*precision;
	void *src = (char*)norm + norm_offset;
        size_t dpitch = flavorTFace*sizeof(float);
        size_t spitch = flavorVolume*sizeof(float);
	hipMemcpy2DAsync(dst, dpitch, src, spitch, flavorTFace*sizeof(float), 2, hipMemcpyDeviceToHost, *stream);
      }
    }
#else
    errorQuda("sendGhost not built on single-GPU build");
#endif

  }


  void cudaColorSpinorField::unpackGhost(void* ghost_spinor, const int dim, 
					 const QudaDirection dir, 
					 const int dagger, hipStream_t* stream) 
  {
    int nFace = (nSpin == 1) ? 3 : 1; //3 faces for asqtad
    int Nint = (nColor * nSpin * 2) / (nSpin == 4 ? 2 : 1);  // (spin proj.) degrees of freedom

    int len = nFace*ghostFace[dim]*Nint;
    int offset = length + ghostOffset[dim]*nColor*nSpin*2;
    offset += (dir == QUDA_BACKWARDS) ? 0 : len;

    void *dst = (char*)v + precision*offset;
    void *src = ghost_spinor;

    hipMemcpyAsync(dst, src, len*precision, hipMemcpyHostToDevice, *stream);
    
    if (precision == QUDA_HALF_PRECISION) {
      int normlen = nFace*ghostFace[dim];
      int norm_offset = stride + ghostNormOffset[dim];
      norm_offset += (dir == QUDA_BACKWARDS) ? 0 : normlen;

      void *dst = (char*)norm + norm_offset*sizeof(float);
      void *src = (char*)ghost_spinor+nFace*Nint*ghostFace[dim]*precision; // norm region of host ghost zone
      hipMemcpyAsync(dst, src, normlen*sizeof(float), hipMemcpyHostToDevice, *stream);
    }

  }

  // Return the location of the field
  QudaFieldLocation cudaColorSpinorField::Location() const { return QUDA_CUDA_FIELD_LOCATION; }

  std::ostream& operator<<(std::ostream &out, const cudaColorSpinorField &a) {
    out << (const ColorSpinorField&)a;
    out << "v = " << a.v << std::endl;
    out << "norm = " << a.norm << std::endl;
    out << "alloc = " << a.alloc << std::endl;
    out << "init = " << a.init << std::endl;
    return out;
  }

} // namespace quda
