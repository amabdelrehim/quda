#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include <quda_internal.h>
#include <color_spinor_field.h>
#include <blas_quda.h>

#include <test_util.h>


// volume per GPU (full lattice dimensions)
const int LX = 16;
const int LY = 16;
const int LZ = 16;
const int LT = 16;
const int Nspin = 4;

// corresponds to 1 iterations for V=16^4, Nspin = 4, at half precision
const int Niter = max(1, 1 * (16*16*16*16*4) / (LX * LY * LZ * LT * Nspin));

const int Nkernels = 31;

cpuColorSpinorField *xH, *yH, *zH, *wH, *vH, *hH, *lH;
cudaColorSpinorField *xD, *yD, *zD, *wD, *vD, *hD, *lD;

void setPrec(ColorSpinorParam &param, const QudaPrecision precision)
{
  param.precision = precision;
  if (Nspin == 1 || precision == QUDA_DOUBLE_PRECISION) {
    param.fieldOrder = QUDA_FLOAT2_FIELD_ORDER;
  } else {
    param.fieldOrder = QUDA_FLOAT4_FIELD_ORDER;
  }
}

void initFields(int prec)
{
  // precisions used for the source field in the copyCuda() benchmark
  QudaPrecision high_aux_prec;
  QudaPrecision low_aux_prec;

  ColorSpinorParam param;
  param.fieldLocation = QUDA_CPU_FIELD_LOCATION;
  param.nColor = 3;
  param.nSpin = Nspin; // =1 for staggered, =2 for coarse Dslash, =4 for 4d spinor
  param.nDim = 4; // number of spacetime dimensions

  param.pad = 0; // padding must be zero for cpu fields
  param.siteSubset = QUDA_PARITY_SITE_SUBSET;
  if (param.siteSubset == QUDA_PARITY_SITE_SUBSET) param.x[0] = LX/2;
  else param.x[0] = LX;
  param.x[1] = LY;
  param.x[2] = LZ;
  param.x[3] = LT;

  param.siteOrder = QUDA_EVEN_ODD_SITE_ORDER;
  param.gammaBasis = QUDA_DEGRAND_ROSSI_GAMMA_BASIS;
  param.precision = QUDA_DOUBLE_PRECISION;
  param.fieldOrder = QUDA_SPACE_SPIN_COLOR_FIELD_ORDER;

  param.create = QUDA_ZERO_FIELD_CREATE;

  vH = new cpuColorSpinorField(param);
  wH = new cpuColorSpinorField(param);
  xH = new cpuColorSpinorField(param);
  yH = new cpuColorSpinorField(param);
  zH = new cpuColorSpinorField(param);
  hH = new cpuColorSpinorField(param);
  lH = new cpuColorSpinorField(param);

  vH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  wH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  xH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  yH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  zH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  hH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);
  lH->Source(QUDA_RANDOM_SOURCE, 0, 0, 0);

  // Now set the parameters for the cuda fields
  param.pad = 0; //LX*LY*LZ/2;
  
  if (param.nSpin == 4) param.gammaBasis = QUDA_UKQCD_GAMMA_BASIS;
  param.fieldLocation = QUDA_CUDA_FIELD_LOCATION;
  param.create = QUDA_ZERO_FIELD_CREATE;

  switch(prec) {
  case 0:
    setPrec(param, QUDA_HALF_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_SINGLE_PRECISION;
    break;
  case 1:
    setPrec(param, QUDA_SINGLE_PRECISION);
    high_aux_prec = QUDA_DOUBLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  case 2:
    setPrec(param, QUDA_DOUBLE_PRECISION);
    high_aux_prec = QUDA_SINGLE_PRECISION;
    low_aux_prec = QUDA_HALF_PRECISION;
    break;
  }

  checkCudaError();

  vD = new cudaColorSpinorField(param);
  wD = new cudaColorSpinorField(param);
  xD = new cudaColorSpinorField(param);
  yD = new cudaColorSpinorField(param);
  zD = new cudaColorSpinorField(param);

  setPrec(param, high_aux_prec);
  hD = new cudaColorSpinorField(param);

  setPrec(param, low_aux_prec);
  lD = new cudaColorSpinorField(param);

  // check for successful allocation
  checkCudaError();

  *vD = *vH;
  *wD = *wH;
  *xD = *xH;
  *yD = *yH;
  *zD = *zH;
  *hD = *hH;
  *lD = *lH;
}


void freeFields()
{

  // release memory
  delete vD;
  delete wD;
  delete xD;
  delete yD;
  delete zD;
  delete hD;
  delete lD;

  // release memory
  delete vH;
  delete wH;
  delete xH;
  delete yH;
  delete zH;
  delete hH;
  delete lH;
}


double benchmark(int kernel, const int niter) {

  double a, b, c;
  quda::Complex a2, b2, c2;

  hipEvent_t start, end;
  hipEventCreate(&start);
  hipEventCreate(&end);
  hipEventRecord(start, 0);

  for (int i=0; i < niter; ++i) {

    switch (kernel) {

    case 0:
      copyCuda(*yD, *hD);
      break;

    case 1:
      copyCuda(*yD, *lD);
      break;
      
    case 2:
      axpbyCuda(a, *xD, b, *yD);
      break;

    case 3:
      xpyCuda(*xD, *yD);
      break;

    case 4:
      axpyCuda(a, *xD, *yD);
      break;

    case 5:
      xpayCuda(*xD, a, *yD);
      break;

    case 6:
      mxpyCuda(*xD, *yD);
      break;

    case 7:
      axCuda(a, *xD);
      break;

    case 8:
      caxpyCuda(a2, *xD, *yD);
      break;

    case 9:
      caxpbyCuda(a2, *xD, b2, *yD);
      break;

    case 10:
      cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
      break;

    case 11:
      axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
      break;

    case 12:
      axpyZpbxCuda(a, *xD, *yD, *zD, b);
      break;

    case 13:
      caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
      break;
      
    case 14:
      cabxpyAxCuda(a, b2, *xD, *yD);
      break;

    case 15:
      caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      break;

    case 16:
      caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      break;

    case 17:
      caxpyXmazCuda(a2, *xD, *yD, *zD);
      break;

      // double
    case 18:
      normCuda(*xD);
      break;

    case 19:
      reDotProductCuda(*xD, *yD);
      break;

    case 20:
      axpyNormCuda(a, *xD, *yD);
      break;

    case 21:
      xmyNormCuda(*xD, *yD);
      break;
      
    case 22:
      caxpyNormCuda(a2, *xD, *yD);
      break;

    case 23:
      caxpyXmazNormXCuda(a2, *xD, *yD, *zD);
      break;

    case 24:
      cabxpyAxNormCuda(a, b2, *xD, *yD);
      break;

    // double2
    case 25:
      cDotProductCuda(*xD, *yD);
      break;

    case 26:
      xpaycDotzyCuda(*xD, a, *yD, *zD);
      break;
      
    case 27:
      caxpyDotzyCuda(a2, *xD, *yD, *zD);
      break;

    // double3
    case 28:
      cDotProductNormACuda(*xD, *yD);
      break;

    case 29:
      cDotProductNormBCuda(*xD, *yD);
      break;

    case 30:
      caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      break;

    default:
      errorQuda("Undefined blas kernel %d\n", kernel);
    }
  }
  
  hipEventRecord(end, 0);
  hipEventSynchronize(end);
  float runTime;
  hipEventElapsedTime(&runTime, start, end);
  hipEventDestroy(start);
  hipEventDestroy(end);

  double secs = runTime / 1000;
  return secs;
}

#define ERROR(a) fabs(norm2(*a##D) - norm2(*a##H)) / norm2(*a##H)

double test(int kernel) {

  double a = 1.5, b = 2.5, c = 3.5;
  quda::Complex a2(a, b), b2(b, -c), c2(a+b, c*a);
  double error = 0;

  switch (kernel) {

  case 0:
    *hD = *hH;
    copyCuda(*yD, *hD);
    yH->copy(*hH);
    error = ERROR(y);
    break;

  case 1:
    *lD = *lH;
    copyCuda(*yD, *lD);
    yH->copy(*lH);
    error = ERROR(y);
    break;
      
  case 2:
    *xD = *xH;
    *yD = *yH;
    axpbyCuda(a, *xD, b, *yD);
    axpbyCpu(a, *xH, b, *yH);
    error = ERROR(y);
    break;

  case 3:
    *xD = *xH;
    *yD = *yH;
    xpyCuda(*xD, *yD);
    xpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 4:
    *xD = *xH;
    *yD = *yH;
    axpyCuda(a, *xD, *yD);
    axpyCpu(a, *xH, *yH);
    error = ERROR(y);
    break;

  case 5:
    *xD = *xH;
    *yD = *yH;
    xpayCuda(*xD, a, *yD);
    xpayCpu(*xH, a, *yH);
    error = ERROR(y);
    break;

  case 6:
    *xD = *xH;
    *yD = *yH;
    mxpyCuda(*xD, *yD);
    mxpyCpu(*xH, *yH);
    error = ERROR(y);
    break;

  case 7:
    *xD = *xH;
    axCuda(a, *xD);
    axCpu(a, *xH);
    error = ERROR(x);
    break;

  case 8:
    *xD = *xH;
    *yD = *yH;
    caxpyCuda(a2, *xD, *yD);
    caxpyCpu(a2, *xH, *yH);
    error = ERROR(y);
    break;

  case 9:
    *xD = *xH;
    *yD = *yH;
    caxpbyCuda(a2, *xD, b2, *yD);
    caxpbyCpu(a2, *xH, b2, *yH);
    error = ERROR(y);
    break;

  case 10:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    cxpaypbzCuda(*xD, a2, *yD, b2, *zD);
    cxpaypbzCpu(*xH, a2, *yH, b2, *zH);
    error = ERROR(z);
    break;

  case 11:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyBzpcxCuda(a, *xD, *yD, b, *zD, c);
    axpyBzpcxCpu(a, *xH, *yH, b, *zH, c);
    error = ERROR(x) + ERROR(y);
    break;

  case 12:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    axpyZpbxCuda(a, *xD, *yD, *zD, b);
    axpyZpbxCpu(a, *xH, *yH, *zH, b);
    error = ERROR(x) + ERROR(y);
    break;

  case 13:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    caxpbypzYmbwCuda(a2, *xD, b2, *yD, *zD, *wD);
    caxpbypzYmbwCpu(a2, *xH, b2, *yH, *zH, *wH);
    error = ERROR(z) + ERROR(y);
    break;
      
  case 14:
    *xD = *xH;
    *yD = *yH;
    cabxpyAxCuda(a, b2, *xD, *yD);
    cabxpyAxCpu(a, b2, *xH, *yH);
    error = ERROR(y) + ERROR(x);
    break;

  case 15:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpbypzCuda(a2, *xD, b2, *yD, *zD);
      caxpbypzCpu(a2, *xH, b2, *yH, *zH);
      error = ERROR(z); }
    break;
    
  case 16:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    {caxpbypczpwCuda(a2, *xD, b2, *yD, c2, *zD, *wD);
      caxpbypczpwCpu(a2, *xH, b2, *yH, c2, *zH, *wH);
      error = ERROR(w); }
    break;

  case 17:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {caxpyXmazCuda(a, *xD, *yD, *zD);
     caxpyXmazCpu(a, *xH, *yH, *zH);
     error = ERROR(y) + ERROR(x);}
    break;

    // double
  case 18:
    *xD = *xH;
    error = fabs(normCuda(*xD) - normCpu(*xH)) / normCpu(*xH);
    break;
    
  case 19:
    *xD = *xH;
    *yD = *yH;
    error = fabs(reDotProductCuda(*xD, *yD) - reDotProductCpu(*xH, *yH)) / fabs(reDotProductCpu(*xH, *yH));
    break;

  case 20:
    *xD = *xH;
    *yD = *yH;
    {double d = axpyNormCuda(a, *xD, *yD);
    double h = axpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 21:
    *xD = *xH;
    *yD = *yH;
    {double d = xmyNormCuda(*xD, *yD);
    double h = xmyNormCpu(*xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;
    
  case 22:
    *xD = *xH;
    *yD = *yH;
    {double d = caxpyNormCuda(a, *xD, *yD);
    double h = caxpyNormCpu(a, *xH, *yH);
    error = ERROR(y) + fabs(d-h)/fabs(h);}
    break;

  case 23:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {double d = caxpyXmazNormXCuda(a, *xD, *yD, *zD);
      double h = caxpyXmazNormXCpu(a, *xH, *yH, *zH);
      error = ERROR(y) + ERROR(x) + fabs(d-h)/fabs(h);}
    break;

  case 24:
    *xD = *xH;
    *yD = *yH;
    {double d = cabxpyAxNormCuda(a, b2, *xD, *yD);
      double h = cabxpyAxNormCpu(a, b2, *xH, *yH);
      error = ERROR(x) + ERROR(y) + fabs(d-h)/fabs(h);}
    break;

    // double2
  case 25:
    *xD = *xH;
    *yD = *yH;
    error = abs(cDotProductCuda(*xD, *yD) - cDotProductCpu(*xH, *yH)) / abs(cDotProductCpu(*xH, *yH));
    break;
    
  case 26:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    { quda::Complex d = xpaycDotzyCuda(*xD, a, *yD, *zD);
      quda::Complex h = xpaycDotzyCpu(*xH, a, *yH, *zH);
      error =  fabs(norm2(*yD) - norm2(*yH)) / norm2(*yH) + abs(d-h)/abs(h);
    }
    break;
    
  case 27:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    {quda::Complex d = caxpyDotzyCuda(a, *xD, *yD, *zD);
      quda::Complex h = caxpyDotzyCpu(a, *xH, *yH, *zH);
    error = ERROR(y) + abs(d-h)/abs(h);}
    break;

    // double3
  case 28:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormACuda(*xD, *yD);
      double3 h = cDotProductNormACpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 29:
    *xD = *xH;
    *yD = *yH;
    { double3 d = cDotProductNormBCuda(*xD, *yD);
      double3 h = cDotProductNormBCpu(*xH, *yH);
      error = fabs(d.x - h.x) / fabs(h.x) + fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;
    
  case 30:
    *xD = *xH;
    *yD = *yH;
    *zD = *zH;
    *wD = *wH;
    *vD = *vH;
    { double3 d = caxpbypzYmbwcDotProductUYNormYCuda(a2, *xD, b2, *yD, *zD, *wD, *vD);
      double3 h = caxpbypzYmbwcDotProductUYNormYCpu(a2, *xH, b2, *yH, *zH, *wH, *vH);
      error = ERROR(z) + ERROR(y) + fabs(d.x - h.x) / fabs(h.x) + 
	fabs(d.y - h.y) / fabs(h.y) + fabs(d.z - h.z) / fabs(h.z); }
    break;

  default:
    errorQuda("Undefined blas kernel %d\n", kernel);
  }

  return error;
}

int main(int argc, char** argv)
{

  int ndim=4, dims[] = {1, 1, 1, 1};
  initCommsQuda(argc, argv, dims, ndim);

  int dev = 0;
  if (argc == 2) dev = atoi(argv[1]);
  initQuda(dev);

  char *names[] = {
    "copyHS",
    "copyLS",
    "axpby",
    "xpy",
    "axpy",
    "xpay",
    "mxpy",
    "ax",
    "caxpy",
    "caxpby",
    "cxpaypbz",
    "axpyBzpcx",
    "axpyZpbx",
    "caxpbypzYmbw",
    "cabxpyAx",
    "caxpbypz",
    "caxpbypczpw",
    "caxpyXmaz",
    "norm",
    "reDotProduct",
    "axpyNorm",
    "xmyNorm",
    "caxpyNorm",
    "caxpyXmazNormX",
    "cabxpyAxNorm",
    "cDotProduct",
    "xpaycDotzy",
    "caxpyDotzy",
    "cDotProductNormA",
    "cDotProductNormB",
    "caxpbypzYmbwcDotProductWYNormY"
  };

  char *prec_str[] = {"half", "single", "double"};
  
  // Only benchmark double precision if supported
#if (__COMPUTE_CAPABILITY__ >= 130)
  int Nprec = 3;
#else
  int Nprec = 2;
#endif

  int niter = Niter;

  // enable the tuning
  quda::setBlasTuning(QUDA_TUNE_YES, QUDA_SILENT);

  for (int prec = 0; prec < Nprec; prec++) {

    printf("\nBenchmarking %s precision with %d iterations...\n\n", prec_str[prec], niter);
    initFields(prec);

    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;

      // do the initial tune
      benchmark(kernel, 1);
    
      // now rerun with more iterations to get accurate speed measurements
      quda::blas_flops = 0;
      quda::blas_bytes = 0;
      
      double secs = benchmark(kernel, 500*niter);
      
      double gflops = (quda::blas_flops*1e-9)/(secs);
      double gbytes = quda::blas_bytes/(secs*1e9);
    
      printf("%-31s: Gflop/s = %6.1f, GB/s = %6.1f\n", names[kernel], gflops, gbytes);
    }
    freeFields();

    // halve the number of iterations for the next precision
    niter /= 2; 
    if (niter==0) niter = 1;
  }

  // clear the error state
  hipGetLastError();

  // lastly check for correctness
  for (int prec = 0; prec < Nprec; prec++) {
    printf("\nTesting %s precision...\n\n", prec_str[prec]);
    initFields(prec);
    
    for (int kernel = 0; kernel < Nkernels; kernel++) {
      // only benchmark "high precision" copyCuda() if double is supported
      if ((Nprec < 3) && (kernel == 0)) continue;
      double error = test(kernel);
      printfQuda("%-35s error = %e, \n", names[kernel], error);
    }
    freeFields();
  }

  endQuda();

  endCommsQuda();
}
